#include "hip/hip_runtime.h"
// autolykos.cu

/*******************************************************************************

    AUTOLYKOS -- Autolukos puzzle cycle

*******************************************************************************/

#include "../include/prehash.h"
#include "../include/mining.h"
#include "../include/reduction.h"
#include "../include/compaction.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

////////////////////////////////////////////////////////////////////////////////
//  Read program input from file
////////////////////////////////////////////////////////////////////////////////
int readInput(
    char * filename,
    void * bound,
    void * mes,
    void * sk,
    void * pk,
    void * x,
    void * w
) {
    FILE * in = fopen(filename, "r");

    int status;

#define SCAN_TO_LITTLE_ENDIAN(x)                                        \
for (int i = 0; i < NUM_SIZE_32 >> 1; ++i)                              \
{                                                                       \
    status = fscanf(                                                    \
        in, "%"SCNx64"\n", (uint64_t *)(x) + (NUM_SIZE_32 >> 1) - i - 1 \
    );                                                                  \
}

#define SCAN_TO_BIG_ENDIAN(x)                                \
for (int i = 0; i < NUM_SIZE_32 >> 1; ++i)                   \
{                                                            \
    status = fscanf(in, "%"SCNx64"\n", (uint64_t *)(x) + i); \
                                                             \
    INPLACE_REVERSE_ENDIAN((uint64_t *)(x) + i);             \
}

    SCAN_TO_LITTLE_ENDIAN(bound);
    SCAN_TO_BIG_ENDIAN(mes);
    SCAN_TO_LITTLE_ENDIAN(sk);

    status = fscanf(in, "%"SCNx8"\n", (uint8_t *)pk);
    SCAN_TO_BIG_ENDIAN((uint8_t *)pk + 1);

    SCAN_TO_LITTLE_ENDIAN(x);

    status = fscanf(in, "%"SCNx8"\n", (uint8_t *)w);
    SCAN_TO_BIG_ENDIAN((uint8_t *)w + 1);

#undef SCAN_TO_BIG_ENDIAN
#undef SCAN_TO_LITTLE_ENDIAN

    fclose(in);

    return status;
}

////////////////////////////////////////////////////////////////////////////////
//  Generate consequtive nonces
////////////////////////////////////////////////////////////////////////////////
__global__ void generate(
    uint64_t * arr,
    uint32_t len,
    uint64_t base
) {
    uint32_t tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint64_t nonce = base + tid;

    INPLACE_REVERSE_ENDIAN(&nonce);

    if (tid < len) arr[tid] = nonce;

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
int main(
    int argc, char ** argv
) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (!deviceCount)
    {
        printf("ABORT: GPU devices did not recognised.");

        return 1;
    }

    //====================================================================//
    //  Host memory
    //====================================================================//
    // hash context
    // (212 + 4) bytes
    blake2b_ctx ctx_h;

    uint32_t bound_h[NUM_SIZE_32];
    uint32_t mes_h[NUM_SIZE_32];
    uint32_t sk_h[NUM_SIZE_32];
    uint8_t pk_h[PK_SIZE_8];
    uint32_t x_h[NUM_SIZE_32];
    uint8_t w_h[PK_SIZE_8];

    if (argc == 1)
    {
        printf("Please, specify the input filename\n");
        fflush(stdout);

        return -1;
    }

    readInput(argv[1], bound_h, mes_h, sk_h, pk_h, x_h, w_h);

    //====================================================================//
    //  Device memory
    //====================================================================//
    // boundary for puzzle
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc((void **)&bound_d, NUM_SIZE_8));

    // nonces
    // H_LEN * L_LEN * NONCE_SIZE_8 bytes // 32 MB
    uint32_t * nonce_d;
    CUDA_CALL(hipMalloc((void **)&nonce_d, H_LEN * L_LEN * NONCE_SIZE_8));

    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, (NUM_SIZE_8 + B_DIM) * 4));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GB
    uint32_t * hash_d;
    CUDA_CALL(hipMalloc((void **)&hash_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (H_LEN * N_LEN * 8 + 4) bytes // ~512 MB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc((void **)&indices_d, H_LEN * N_LEN * 8 + 4));

    // potential solutions of puzzle
    // H_LEN * L_LEN * 4 * 8 bytes // 16 * 8 MB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, H_LEN * L_LEN * NUM_SIZE_8));

    //====================================================================//
    //  Random generator initialization
    //====================================================================//
    /// original /// hiprandGenerator_t gen;
    /// original /// CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
    /// original /// 
    /// original /// time_t rawtime;
    /// original /// // get current time (ms)
    /// original /// time(&rawtime);

    /// original /// // set seed
    /// original /// CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (uint64_t)rawtime));

    //====================================================================//
    //  Memory: Host -> Device
    //====================================================================//
    // bound
    CUDA_CALL(hipMemcpy(
        (void *)bound_d, (void *)bound_h, NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    // public key
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)pk_h, PK_SIZE_8, hipMemcpyHostToDevice
    ));

    // message
    CUDA_CALL(hipMemcpy(
        (void *)((uint8_t *)data_d + PK_SIZE_8), (void *)mes_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // secret key
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + PK2_SIZE_32 + 2 * NUM_SIZE_32), (void *)sk_h,
        NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    uint32_t ind = 0;
    uint32_t is_first = 1;
    int i;
    struct timeval t1, t2, t3;
    uint64_t base = 0;

    for (i = 0; !ind && i < 1; ++i)
    {
        gettimeofday(&t1, 0);

        // on obtaining solution
        if (is_first)
        {
            // one time secret key: host -> device
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + PK2_SIZE_32 + NUM_SIZE_32), (void *)x_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // one time public key: host -> device
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8),
                (void *)w_h, PK_SIZE_8, hipMemcpyHostToDevice
            ));

            // precalculate hashes
            prehash(data_d, hash_d, indices_d);

            is_first = 0;
        }

        CUDA_CALL(hipDeviceSynchronize());
        gettimeofday(&t2, 0);
        /// prehash /// break;

        // generate nonces
        /// original /// CURAND_CALL(hiprandGenerate(gen, nonce_d, H_LEN * L_LEN * NONCE_SIZE_8));
        generate<<<1 + (H_LEN * L_LEN - 1) / B_DIM, B_DIM>>>(
            (uint64_t *)nonce_d, N_LEN, base
        );
        base += H_LEN * L_LEN;

        // calculate unfinalized hash of message
        initMining(&ctx_h, mes_h, NUM_SIZE_8);

        // context: host -> device
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + PK2_SIZE_32 + 3 * NUM_SIZE_32), (void *)&ctx_h,
            sizeof(blake2b_ctx), hipMemcpyHostToDevice
        ));

        // calculate solution candidates
        blockMining<<<1 + (L_LEN - 1) / B_DIM, B_DIM>>>(
            bound_d, data_d, nonce_d, hash_d, res_d, indices_d
        );

        // try to find solution
        ind = findNonZero(indices_d, indices_d + H_LEN * L_LEN, H_LEN * L_LEN);
    }

    hipDeviceSynchronize();
    gettimeofday(&t3, 0);

    //====================================================================//
    //  Time evaluation
    //====================================================================//
    double time
        = (1000000. * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec)
        / 1000000.0;
    printf("Prehash time: %.5f (s) \n", time);

    time = (1000000. * (t3.tv_sec - t2.tv_sec) + t3.tv_usec - t2.tv_usec)
        / 1000000.0;
    printf("Mining time: %.5f (s) \n", time);

    fflush(stdout);

    //====================================================================//
    //  [DEBUG] Result with index
    //====================================================================//
    uint32_t * res_h = (uint32_t *)malloc(H_LEN * L_LEN * NUM_SIZE_8);

    CUDA_CALL(hipMemcpy(
        (void *)res_h, (void *)res_d, H_LEN * L_LEN * NUM_SIZE_8,
        hipMemcpyDeviceToHost
    ));

    uint32_t * nonce_h = (uint32_t *)malloc(H_LEN * L_LEN * NONCE_SIZE_8);

    CUDA_CALL(hipMemcpy(
        (void *)nonce_h, (void *)nonce_d, H_LEN * L_LEN * NONCE_SIZE_8,
        hipMemcpyDeviceToHost
    ));

    if (ind)
    {
        printf("iteration = %d, index = %d\n", i - 1, ind - 1);

        printf(
            "m     = 0x%016lX %016lX %016lX %016lX\n",
            ((uint64_t *)mes_h)[3], ((uint64_t *)mes_h)[2],
            ((uint64_t *)mes_h)[1], ((uint64_t *)mes_h)[0]
        );

        printf(
            "pk    = 0x%02lX %016lX %016lX %016lX %016lX\n",
            ((uint8_t *)pk_h)[0],
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 0),
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 1),
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 2),
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 3)
        );

        printf(
            "w     = 0x%02lX %016lX %016lX %016lX %016lX\n",
            ((uint8_t *)w_h)[0],
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 0),
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 1),
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 2),
            REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 3)
        );

        printf(
            "nonce = 0x%016lX\n",
            REVERSE_ENDIAN(((uint64_t *)nonce_h) + ind - 1)
        );

        printf(
            "d     = 0x%016lX %016lX %016lX %016lX\n",
            ((uint64_t *)res_h)[(ind - 1) * 4 + 3],
            ((uint64_t *)res_h)[(ind - 1) * 4 + 2],
            ((uint64_t *)res_h)[(ind - 1) * 4 + 1],
            ((uint64_t *)res_h)[(ind - 1) * 4]
        );

        fflush(stdout);
    }

    free(res_h);
    free(nonce_h);

    //====================================================================//
    //  Free device memory
    //====================================================================//
    /// original /// CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(bound_d));
    CUDA_CALL(hipFree(nonce_d));
    CUDA_CALL(hipFree(hash_d));
    CUDA_CALL(hipFree(data_d));
    CUDA_CALL(hipFree(indices_d));
    CUDA_CALL(hipFree(res_d));

    return 0;
}

// autolykos.cu
