#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "blake2b.h"
#include "autolykos.h"

// L = GS * BS
#define GS 15625
#define BS 64

#define CUDA_CALL(x) do { if((x) != hipSuccess) {  \
printf("Error at %s:%d\n",__FILE__,__LINE__);       \
return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) {  \
printf("Error at %s:%d\n",__FILE__,__LINE__);                   \
return EXIT_FAILURE;}} while(0)

void initRand(
    hiprandGenerator_t * gen,
    uint32_t ** non
) {
    CURAND_CALL(hiprandCreateGenerator(gen, HIPRAND_RNG_PSEUDO_MTGP32));
    CUDA_CALL(hipMalloc((void **)non, L * sizeof(uint32_t)));

    time_t rawtime;
    time(&rawtime);
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (uint64_t)rawtime));

    return;
}

int main(int argc, char ** argv)
{
    hiprandGenerator_t gen;
    uint32_t * non;

    initRand(&gen, &non);

    blake2b_ctx * ctx;
    hipMalloc(&ctx, L * sizeof(blake2b_ctx));

    while(1)
    {
        // 

        // generate nonces
        CURAND_CALL(hiprandGenerate(gen, non, L));

        // calculate hashes
        blockMining<<<GS, BS>>>(
            // context
            ctx,
            // optional secret key
            NULL, 0,
            // message
            in, 64,
            // pregenerated nonces
            non,
            // hashes
            out, 32
        );

        findSolution<<<GS, BS>>>(
        );
    }

    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(non));
    CUDA_CALL(hipFree(ctx));

    return 0;
}

