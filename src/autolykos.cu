#include "hip/hip_runtime.h"
// autolykos.cu

#include "../include/prehash.h"
#include "../include/mining.h"
#include "../include/reduction.h"
#include "../include/compaction.h"
#include "../include/curve25519-donna.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

// consequtive nonces
__global__ void generate(
    uint64_t * arr,
    uint32_t len,
    uint64_t base
) {
    uint32_t tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < len) arr[tid] = base + tid;

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    //====================================================================//
    //  Host memory
    //====================================================================//
    static const uint8_t basepoint[NUM_SIZE_8] = {9};

    // hash context
    // (212 + 4) bytes
    blake2b_ctx ctx_h;

    // message stub
    // NUM_SIZE_8 bytes
    uint32_t mes_h[NUM_SIZE_32] = {0, 0, 0, 0, 0, 0, 0, 1}; 

    // generate secret key
    uint32_t sk_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 1, 2}; 

    ((uint8_t *)sk_h)[0] &= 248;
    ((uint8_t *)sk_h)[31] &= 127;
    ((uint8_t *)sk_h)[31] |= 64;

    // generate public key
    /// stub /// uint32_t pk_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 3, 4}; 
    uint32_t pk_h[NUM_SIZE_32];
    curve25519_donna((uint8_t *)pk_h, (uint8_t *)sk_h, basepoint);

    printf("Public key generated\n");
    fflush(stdout);

    // generate one time secret key
    uint32_t x_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 5, 6}; 

    ((uint8_t *)x_h)[0] &= 248;
    ((uint8_t *)x_h)[31] &= 127;
    ((uint8_t *)x_h)[31] |= 64;

    // generate one time public key
    /// stub /// uint32_t w_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 7, 8}; 
    uint32_t w_h[NUM_SIZE_32];
    curve25519_donna((uint8_t *)w_h, (uint8_t *)x_h, basepoint);

    printf("One-time public key generated\n");
    fflush(stdout);

    //====================================================================//
    //  Device memory
    //====================================================================//
    // nonces
    // H_LEN * L_LEN * NONCE_SIZE_8 bytes // 32 MB
    uint32_t * non_d;
    CUDA_CALL(hipMalloc((void **)&non_d, H_LEN * L_LEN * NONCE_SIZE_8));

    // data: pk || mes || w || x || sk || ctx
    // (5 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, (NUM_SIZE_8 + B_DIM) * 4));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GB
    uint32_t * hash_d;
    CUDA_CALL(hipMalloc((void **)&hash_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (H_LEN * N_LEN * 8 + 4) bytes // ~512 MB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc((void **)&indices_d, H_LEN * N_LEN * 8 + 4));

    /// original /// // potential solutions of puzzle
    /// original /// // H_LEN * L_LEN * 4 bytes // 16 MB
    /// original /// uint32_t * res_d;
    /// original /// CUDA_CALL(hipMalloc((void **)&res_d, (uint32_t)H_LEN * L_LEN * 4));

    // potential solutions of puzzle
    // H_LEN * L_LEN * 4 * 8 bytes // 16 * 8 MB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, H_LEN * L_LEN * 4 * 8));

    //====================================================================//
    //  Random generator initialization
    //====================================================================//
    /// original /// hiprandGenerator_t gen;
    /// original /// CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
    /// original /// 
    /// original /// time_t rawtime;
    /// original /// // get current time (ms)
    /// original /// time(&rawtime);

    /// original /// // set seed
    /// original /// CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (uint64_t)rawtime));

    //====================================================================//
    //  Memory: Host -> Device
    //====================================================================//
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)pk_h, NUM_SIZE_8, hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + NUM_SIZE_32), (void *)mes_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 2 * NUM_SIZE_32), (void *)w_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 3 * NUM_SIZE_32), (void *)x_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 4 * NUM_SIZE_32), (void *)sk_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    uint32_t ind = 0;
    uint32_t is_first = 1;
    int i;
    struct timeval t1, t2;
    uint64_t base = 0;

    for (i = 0; !ind && i < 24; ++i) //>>>(1)
    {
        /// prehash /// gettimeofday(&t1, 0);

        // on obtaining solution
        if (is_first)
        {
            //>>>genSKey();
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + 3 * NUM_SIZE_32), (void *)x_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));
            //>>>genPKey();
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + 2 * NUM_SIZE_32), (void *)w_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            prehash(data_d, hash_d, indices_d);

            is_first = 0;

            gettimeofday(&t1, 0);
        }

        /// prehash /// CUDA_CALL(hipDeviceSynchronize());
        /// prehash /// gettimeofday(&t2, 0);
        /// prehash /// break;

        // generate nonces
        /// original /// CURAND_CALL(hiprandGenerate(gen, non_d, H_LEN * L_LEN * NONCE_SIZE_8));
        generate<<<1 + (H_LEN * L_LEN - 1) / B_DIM, B_DIM>>>(
            (uint64_t *)non_d, N_LEN, base
        );
        base += H_LEN * L_LEN;

        // calculate unfinalized hash of message
        initMining(&ctx_h, mes_h, NUM_SIZE_8);

        // context: host -> device
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + 5 * NUM_SIZE_32), (void *)&ctx_h,
            sizeof(blake2b_ctx), hipMemcpyHostToDevice
        ));

        // calculate hashes
        blockMining<<<1 + (L_LEN - 1) / B_DIM, B_DIM>>>(
            data_d, non_d, hash_d, res_d, indices_d
        );

        // try to find solution
        ind = findNonZero(indices_d, indices_d + H_LEN * L_LEN);
    }

    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    //====================================================================//
    //  Time evaluation
    //====================================================================//
    double time
        = (1000000. * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec)
        / 1000000.0;
    printf("Time to generate: %.5f (s) \n", time);

    //====================================================================//
    //  [DEBUG] Result with index
    //====================================================================//
    uint32_t * res_h = (uint32_t *)malloc(H_LEN * L_LEN * 4 * 8);

    CUDA_CALL(hipMemcpy(
        (void *)res_h, (void *)res_d, H_LEN * L_LEN * 4 * 8,
        hipMemcpyDeviceToHost
    ));

    if (ind)
    {
        printf("ind = %d, i = %d\n", ind - 1, i - 1);

        printf(
            "0x%016lX %016lX %016lX %016lX\n",
            ((uint64_t *)res_h)[(ind - 1) * 4 + 3],
            ((uint64_t *)res_h)[(ind - 1) * 4 + 2],
            ((uint64_t *)res_h)[(ind - 1) * 4 + 1],
            ((uint64_t *)res_h)[(ind - 1) * 4]
        );

        fflush(stdout);
    }

    free(res_h);

    //====================================================================//
    //  Free device memory
    //====================================================================//
    /// original /// CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(non_d));
    CUDA_CALL(hipFree(hash_d));
    CUDA_CALL(hipFree(data_d));
    CUDA_CALL(hipFree(indices_d));
    CUDA_CALL(hipFree(res_d));

    return 0;
}

// autolykos.cu
