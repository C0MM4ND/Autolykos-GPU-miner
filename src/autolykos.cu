#include "hip/hip_runtime.h"
// autolykos.cu

#include "../include/prehash.h"
#include "../include/validation.h"
#include "../include/reduction.h"
#include "../include/compaction.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

__global__ void generate(
    uint64_t * arr,
    uint32_t len,
    uint64_t base
) {
    uint32_t tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < len) arr[tid] = base + tid;

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    //====================================================================//
    //  Host memory
    //====================================================================//
    // hash context
    // (212 + 4) bytes
    blake2b_ctx ctx_h;

    // message stub
    // NUM_SIZE_8 bytes
    uint32_t mes_h[NUM_SIZE_32] = {0, 0, 0, 0, 0, 0, 0, 1}; 

    // secret key
    //>>>genSKey();
    uint32_t sk_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 1, 2}; 

    // public key
    //>>>genPKey();
    uint32_t pk_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 3, 4}; 

    // one time secret key
    //>>>genSKey();
    uint32_t x_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 5, 6}; 

    // one time public key
    //>>>genPKey();
    uint32_t w_h[NUM_SIZE_32] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 7, 8}; 

    //====================================================================//
    //  Device memory
    //====================================================================//
    // nonces
    // H_LEN * L_LEN * NONCE_SIZE_8 bytes // 32 MB
    uint32_t * non_d;
    CUDA_CALL(hipMalloc((void **)&non_d, H_LEN * L_LEN * NONCE_SIZE_8));

    // data: pk || mes || w || x || sk || ctx
    // (5 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, (NUM_SIZE_8 + B_DIM) * 4));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GB
    uint32_t * hash_d;
    CUDA_CALL(hipMalloc((void **)&hash_d, (uint32_t)N_LEN * NUM_SIZE_8));

    /// debug /// uint32_t * hash_dd;
    /// debug /// CUDA_CALL(hipMalloc((void **)&hash_dd, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (H_LEN * N_LEN * 8 + 4) bytes // ~512 MB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc((void **)&indices_d, H_LEN * N_LEN * 8 + 4));

    /// original /// // potential solutions of puzzle
    /// original /// // H_LEN * L_LEN * 4 bytes // 16 MB
    /// original /// uint32_t * res_d;
    /// original /// CUDA_CALL(hipMalloc((void **)&res_d, (uint32_t)H_LEN * L_LEN * 4));

    // potential solutions of puzzle
    // H_LEN * L_LEN * 4 * 8 bytes // 16 * 8 MB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, H_LEN * L_LEN * 4 * 8));

    //====================================================================//
    //  Random generator initialization
    //====================================================================//
    /// original /// hiprandGenerator_t gen;
    /// original /// CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
    /// original /// 
    /// original /// time_t rawtime;
    /// original /// // get current time (ms)
    /// original /// time(&rawtime);

    /// original /// // set seed
    /// original /// CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (uint64_t)rawtime));

    //====================================================================//
    //  Memory: Host -> Device
    //====================================================================//
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)pk_h, NUM_SIZE_8, hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + NUM_SIZE_32), (void *)mes_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 2 * NUM_SIZE_32), (void *)w_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 3 * NUM_SIZE_32), (void *)x_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 4 * NUM_SIZE_32), (void *)sk_h, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    uint32_t ind = 0;
    uint32_t is_first = 1;
    int i;
    struct timeval t1, t2;
    uint64_t base = 0;

    for (i = 0; !ind && i < 24000; ++i) //>>>(1)
    {
        /// prehash /// gettimeofday(&t1, 0);

        // on obtaining solution
        if (is_first)
        {
            //>>>genSKey();
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + 3 * NUM_SIZE_32), (void *)x_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));
            //>>>genPKey();
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + 2 * NUM_SIZE_32), (void *)w_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            prehash(data_d, hash_d, indices_d);
            /// debug /// prehash(data_d, hash_dd, indices_d);

            is_first = 0;

            gettimeofday(&t1, 0);
        }

        /// prehash /// CUDA_CALL(hipDeviceSynchronize());
        /// prehash /// gettimeofday(&t2, 0);
        /// prehash /// break;

        // generate nonces
        /// original /// CURAND_CALL(hiprandGenerate(gen, non_d, H_LEN * L_LEN * NONCE_SIZE_8));
        generate<<<1 + (H_LEN * L_LEN - 1) / B_DIM, B_DIM>>>(
            (uint64_t *)non_d, N_LEN, base
        );
        base += H_LEN * L_LEN;

        // calculate unfinalized hash of message
        initMining(&ctx_h, mes_h, NUM_SIZE_8);

        // context: host -> device
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + 5 * NUM_SIZE_32), (void *)&ctx_h,
            sizeof(blake2b_ctx), hipMemcpyHostToDevice
        ));

        // calculate hashes
        blockMining<<<1 + (L_LEN - 1) / B_DIM, B_DIM>>>(
            data_d, non_d, hash_d, res_d, indices_d
        );

        // try to find solution
        ind = findNonZero(indices_d, indices_d + H_LEN * L_LEN);

        /// debug /// printf("%d ", ind);
        /// debug /// fflush(stdout);

        /// debug /// ind = 0;

        /// debug /// uint32_t * indices_h = (uint32_t *)malloc(H_LEN * L_LEN * 4);

        /// debug /// CUDA_CALL(hipMemcpy(
        /// debug ///     (void *)indices_h, (void *)res_d,
        /// debug ///     H_LEN * L_LEN * 4, hipMemcpyDeviceToHost
        /// debug /// ));

        /// debug /// int k = 0;
        /// debug /// for (int i = 0; i < H_LEN * L_LEN; ++i)
        /// debug /// {
        /// debug ///     if (indices_h[i] > 0)
        /// debug ///     {
        /// debug ///         printf("%d\n", indices_h[i]);
        /// debug ///     }
        /// debug ///     else
        /// debug ///     {
        /// debug ///         ++k;
        /// debug ///     }
        /// debug /// }
        /// debug /// printf("%d %d\n", k, H_LEN * L_LEN);

        /// debug /// free(indices_h);
    }

    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    //====================================================================//
    //  Time evaluation
    //====================================================================//
    double time
        = (1000000. * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec)
        / 1000000.0;
    printf("Time to generate: %.5f (s) \n", time);

    //====================================================================//
    //  [DEBUG] Result in index
    //====================================================================//
    uint32_t * res_h = (uint32_t *)malloc(H_LEN * L_LEN * 4 * 8);

    CUDA_CALL(hipMemcpy(
        (void *)res_h, (void *)res_d, H_LEN * L_LEN * 4 * 8,
        hipMemcpyDeviceToHost
    ));

    /// debug /// uint32_t * res_h = (uint32_t *)malloc((uint32_t)N_LEN * NUM_SIZE_8);
    /// debug /// uint32_t * res_hh = (uint32_t *)malloc((uint32_t)N_LEN * NUM_SIZE_8);

    /// debug /// CUDA_CALL(hipMemcpy(
    /// debug ///     (void *)res_h, (void *)hash_d, (uint32_t)N_LEN * NUM_SIZE_8,
    /// debug ///     hipMemcpyDeviceToHost
    /// debug /// ));
    /// debug /// CUDA_CALL(hipMemcpy(
    /// debug ///     (void *)res_hh, (void *)hash_dd, (uint32_t)N_LEN * NUM_SIZE_8,
    /// debug ///     hipMemcpyDeviceToHost
    /// debug /// ));

    /// debug /// for (uint32_t i = 0; i < N_LEN * NUM_SIZE_32; ++i)
    /// debug /// {
    /// debug ///     if (res_h[i] != res_hh[i])
    /// debug ///         printf("ERROR");
    /// debug /// }

    if (ind)
    {
        printf("ind = %d, i = %d\n", ind - 1, i - 1);
        printf(
            "0x%016lX %016lX %016lX %016lX\n",
            ((uint64_t *)res_h)[(ind - 1) * 4 + 3],
            ((uint64_t *)res_h)[(ind - 1) * 4 + 2],
            ((uint64_t *)res_h)[(ind - 1) * 4 + 1],
            ((uint64_t *)res_h)[(ind - 1) * 4]
        );
        fflush(stdout);
    }

    /// debug /// for (int i = 0; i < H_LEN * L_LEN; ++i)
    /// debug /// {
    /// debug ///     printf("%d ", res_h[i]);
    /// debug /// }

    free(res_h);

    //====================================================================//
    //  Free device memory
    //====================================================================//
    /// original /// CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(non_d));
    CUDA_CALL(hipFree(hash_d));
    CUDA_CALL(hipFree(data_d));
    CUDA_CALL(hipFree(indices_d));
    CUDA_CALL(hipFree(res_d));

    return 0;
}

// autolykos.cu
