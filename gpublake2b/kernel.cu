#include "hip/hip_runtime.h"
#include "blake2b.h"

// Cyclic right rotation.
#ifndef ROTR64
#define ROTR64(x, y)  (((x) >> (y)) ^ ((x) << (64 - (y))))
#endif

// Little-endian byte access.
#define B2B_GET64(p)                            \
    (((uint64_t) ((uint8_t *) (p))[0]) ^        \
    (((uint64_t) ((uint8_t *) (p))[1]) << 8) ^  \
    (((uint64_t) ((uint8_t *) (p))[2]) << 16) ^ \
    (((uint64_t) ((uint8_t *) (p))[3]) << 24) ^ \
    (((uint64_t) ((uint8_t *) (p))[4]) << 32) ^ \
    (((uint64_t) ((uint8_t *) (p))[5]) << 40) ^ \
    (((uint64_t) ((uint8_t *) (p))[6]) << 48) ^ \
    (((uint64_t) ((uint8_t *) (p))[7]) << 56))

// G Mixing function.
#define B2B_G(a, b, c, d, x, y)     \
{                                   \
    v[a] = v[a] + v[b] + x;         \
    v[d] = ROTR64(v[d] ^ v[a], 32); \
    v[c] = v[c] + v[d];             \
    v[b] = ROTR64(v[b] ^ v[c], 24); \
    v[a] = v[a] + v[b] + y;         \
    v[d] = ROTR64(v[d] ^ v[a], 16); \
    v[c] = v[c] + v[d];             \
    v[b] = ROTR64(v[b] ^ v[c], 63); \
}

////////////////////////////////////////////////////////////////////////////////
// Hash
////////////////////////////////////////////////////////////////////////////////
__global__ void blake2b(
    blake2b_ctx * ctx,
    void * out,
    uint32_t outlen,
    const void * key,
    uint32_t keylen,
    const void * in,
    uint32_t inlen
) {
    int k;

    const uint64_t blake2b_iv[8] = {
        0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
        0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
        0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
        0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
    };

    const uint8_t sigma[12][16] = {
        { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
        { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
        { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
        { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
        { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
        { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
        { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
        { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
        { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
        { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
        { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
        { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
    };

    //====================================================================//
#pragma unroll
    for (k = 0; k < 8; ++k)
    {
        ctx->h[k] = blake2b_iv[k];
    }

    ctx->h[0] ^= 0x01010000 ^ (keylen << 8) ^ outlen;

    ctx->t[0] = 0;
    ctx->t[1] = 0;
    ctx->c = 0;
    ctx->outlen = outlen;

#pragma unroll
    for (k = keylen; k < 128; ++k)
    {
        ctx->b[k] = 0;
    }

    //====================================================================//
    int i;
    uint64_t v[16];
    uint64_t m[16];

    for (k = 0; k < keylen & 0xFFFFFF80; ++k)
    {
        while (ctx->c < 128)
        {
            ctx->b[ctx->c++] = ((const uint8_t *)key)[k++];
        }

        ctx->t[0] += ctx->c;
        ctx->t[1] += 1 - !(ctx->t[0] < ctx->c);

#pragma unroll
        for (i = 0; i < 8; ++i)
        {
            v[i] = ctx->h[i];
            v[i + 8] = blake2b_iv[i];
        }

        v[12] ^= ctx->t[0];
        v[13] ^= ctx->t[1];

#pragma unroll
        for (i = 0; i < 16; i++)
        {
            m[i] = B2B_GET64(&ctx->b[8 * i]);
        }

#pragma unroll
        for (i = 0; i < 12; ++i)
        {
            B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
            B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
            B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
            B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
            B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
            B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
            B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
            B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
        }

#pragma unroll
        for (i = 0; i < 8; ++i)
        {
            ctx->h[i] ^= v[i] ^ v[i + 8];
        }

        ctx->c = 0;

        ctx->b[ctx->c++] = ((const uint8_t *)key)[k];
    }

    while (k < keylen)
    {
        ctx->b[ctx->c++] = ((const uint8_t *)key)[k++];
    }

    //ctx->c = (keylen > 0)? 128: ctx->c;
    ctx->c = ((1 - !(keylen > 0)) << 7) + (!(keylen > 0)) * ctx->c;

    //====================================================================//
    for (k = 0; ctx->c < 128 && k < inlen; ++k)
    {
        ctx->b[ctx->c++] = ((const uint8_t *)in)[k];
    }

    while (k < inlen)
    {
        ctx->t[0] += ctx->c;
        ctx->t[1] += 1 - !(ctx->t[0] < ctx->c);

#pragma unroll
        for (i = 0; i < 8; ++i)
        {
            v[i] = ctx->h[i];
            v[i + 8] = blake2b_iv[i];
        }

        v[12] ^= ctx->t[0];
        v[13] ^= ctx->t[1];

#pragma unroll
        for (i = 0; i < 16; i++)
        {
            m[i] = B2B_GET64(&ctx->b[8 * i]);
        }

#pragma unroll
        for (i = 0; i < 12; ++i)
        {
            B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
            B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
            B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
            B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
            B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
            B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
            B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
            B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
        }

#pragma unroll
        for (i = 0; i < 8; ++i)
        {
            ctx->h[i] ^= v[i] ^ v[i + 8];
        }

        ctx->c = 0;
       
        while (ctx->c < 128 && k < inlen)
        {
            ctx->b[ctx->c++] = ((const uint8_t *)in)[k++];
        }
    }

    //====================================================================//
    ctx->t[0] += ctx->c;
    ctx->t[1] += 1 - !(ctx->t[0] < ctx->c);

    while (ctx->c < 128)
    {
        ctx->b[ctx->c++] = 0;
    }

#pragma unroll
    for (i = 0; i < 8; ++i)
    {
        v[i] = ctx->h[i];
        v[i + 8] = blake2b_iv[i];
    }

    v[12] ^= ctx->t[0];
    v[13] ^= ctx->t[1];
    v[14] = ~v[14];

#pragma unroll
    for (i = 0; i < 16; i++)
    {
        m[i] = B2B_GET64(&ctx->b[8 * i]);
    }

#pragma unroll
    for (i = 0; i < 12; ++i)
    {
        B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
        B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
        B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
        B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
        B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
        B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
        B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
        B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
    }

#pragma unroll
    for (i = 0; i < 8; ++i)
    {
        ctx->h[i] ^= v[i] ^ v[i + 8];
    }

    for (k = 0; k < ctx->outlen; ++k)
    {
        ((uint8_t *)out)[k] = (ctx->h[k >> 3] >> ((k & 7) << 3)) & 0xFF;
    }

    return;
}
