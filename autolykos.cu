// autolykos.cu

#include "autolykos.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    //====================================================================//
    //  Host memory
    //====================================================================//
    int ind = 0;

    // BLAKE_2B_256 params
    // 64 bytes
    const uint64_t blake2b_iv[8] = {
        0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
        0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
        0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
        0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
    };

    // pemutations of {0, 1, ..., 15}
    // 192 bytes
    const uint8_t sigma[192] = {
        0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15,
        14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3,
        11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4,
        7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8,
        9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13,
        2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9,
        12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11,
        13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10,
        6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5,
        10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0,
        0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15,
        14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3
    };

    // 212 bytes
    blake2b_ctx ctx_h;

    // 8 * 32 bits = 32 bytes
    uint32_t mes_h[8] = {0, 0, 0, 0, 0, 0, 0, 0}; 

    // L_LEN * 256 bits
    uint32_t * res_h = (uint32_t *)malloc(L_LEN * 8 * 4); 

    //====================================================================//
    //  Device memory
    //====================================================================//
    // nonces
    // 4 * L_LEN * H_LEN bytes
    uint32_t * non_d;
    CUDA_CALL(hipMalloc((void **)&non_d, 4 * L_LEN * H_LEN));

    // data: blake2b_iv || sigma || sk || pk || mes || w || x
    // (256 + 5 * NUM_BYTE_SIZE) bytes
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, 256 + 5 * NUM_BYTE_SIZE));

    // precalculated hashes
    // NUM_BYTE_SIZE * N_LEN bytes
    uint32_t * hash_d;
    CUDA_CALL(hipMalloc((void **)&hash_d, NUM_BYTE_SIZE * N_LEN));

    // indices of unfinalized hashes
    // 4 * H_LEN * N_LEN bytes
    uint32_t * unfinalized_d;
    CUDA_CALL(hipMalloc((void **)&unfinalized_d, 8 * H_LEN * N_LEN));

    // 4 * H_LEN * N_LEN bytes
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, 4 * H_LEN * N_LEN));

    //====================================================================//
    //  Random generator initialization
    //====================================================================//
    // intialize random generator
    hiprandGenerator_t gen;

    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
    
    time_t rawtime;
    time(&rawtime);
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (uint64_t)rawtime));

    //====================================================================//
    /// debug /// uint32_t arr_h[0x4000000];

    /// debug /// for (int i = 0; i < 0x4000000; ++i)
    /// debug /// {
    /// debug ///     arr_h[i] = 0;
    /// debug /// }

    /// debug /// for (int i = 13; i < 0x4000000; i += 7)
    /// debug /// {
    /// debug ///     arr_h[i] = i;
    /// debug /// }

    /// debug /// uint32_t * in_d;
    /// debug /// uint32_t * out_d;

    /// debug /// CUDA_CALL(hipMalloc((void **)&in_d, 0x4000000 * 4));
    /// debug /// CUDA_CALL(hipMalloc((void **)&out_d, 0x4000000 * 2));

    /// debug /// CUDA_CALL(hipMemcpy(
    /// debug ///     (void *)in_d, arr_h, 0x4000000 * 4, hipMemcpyHostToDevice
    /// debug /// ));

    /// debug /// printf("%d\n", findNonZero(in_d, out_d));

    /// debug /// CUDA_CALL(hipFree(in_d));
    /// debug /// CUDA_CALL(hipFree(out_d));

    //====================================================================//
    // secret key
    //>>>genSKey();
    uint32_t sk_h[8] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 1, 2}; 
    //>>>genPKey();
    uint32_t pk_h[8] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 3, 4}; 

    //====================================================================//
    //  Memory: Host -> Device
    //====================================================================//
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)blake2b_iv, 64, hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 16), (void *)sigma, 192, hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 64), (void)sk_h, NUM_BYTE_SIZE, hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 64 + (NUM_BYTE_SIZE >> 2)), (void)pk_h, NUM_BYTE_SIZE, hipMemcpyHostToDevice
    ));
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + 64 + 2 * (NUM_BYTE_SIZE >> 2)), (void)mes_h, NUM_BYTE_SIZE, hipMemcpyHostToDevice
    ));

    // one time secret key
    uint32_t x_h[8] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 5, 6}; 
    //>>>genPKey();
    uint32_t w_h[8] = {0xA, 0xB, 0xC, 0xD, 0xE, 0xF, 7, 8}; 

    while (ind) //>>>(1)
    {
        if (ind)
        {
            //>>>genSKey();
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + 64 + 4 * (NUM_BYTE_SIZE >> 2)), (void)x_h, NUM_BYTE_SIZE,
                hipMemcpyHostToDevice
            ));
            //>>>genPKey();
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + 3 * (NUM_BYTE_SIZE >> 2)), (void)w_h, NUM_BYTE_SIZE,
                hipMemcpyHostToDevice
            ));

            initPrehash<<<1 + (N_LEN - 1) / B_DIM, B_DIM>>>(data_d, hash_d, unfinalized_d);
            //>>>updatePrehash(data_d, hash_d, unfinalized_d);
            finalizePrehash<<<1 + (N_LEN - 1) / B_DIM, B_DIM>>>(data_d, hash_d);
        }

        // generate nonces
        CURAND_CALL(hiprandGenerate(gen, non_d, 4 * L_LEN * H_LEN));

        // calculate unfinalized hash of message
        initMining(ctx_h, sk_h, mes_h, NUM_BYTE_SIZE);

        // context: host -> device
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + 5 * (NUM_BYTE_SIZE >> 2)),
            (void *)ctx_h, sizeof(blake2b_ctx), hipMemcpyHostToDevice
        ));

        // calculate hashes
        blockMining<<<G_DIM, B_DIM>>>(ctx_d, non_d, hash_d, res_d, unfinalized_d);
        ind = findNonZero(unfinalized_d, out_d + 4 * H_LEN * N_LEN);
    }

    //====================================================================//
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(non_d));
    CUDA_CALL(hipFree(res_d));
    CUDA_CALL(hipFree(unfinalized_d));
    CUDA_CALL(hipFree(hash_d));
    CUDA_CALL(hipFree(data_d));

    return 0;
}

// autolykos.cu
