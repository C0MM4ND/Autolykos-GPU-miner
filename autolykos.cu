#include "hip/hip_runtime.h"
#include "autolykos.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>

// Little-endian byte access
#ifndef B2B_GET64
#define B2B_GET64(p)                            \
    (((uint64_t) ((uint8_t *) (p))[0]) ^        \
    (((uint64_t) ((uint8_t *) (p))[1]) << 8) ^  \
    (((uint64_t) ((uint8_t *) (p))[2]) << 16) ^ \
    (((uint64_t) ((uint8_t *) (p))[3]) << 24) ^ \
    (((uint64_t) ((uint8_t *) (p))[4]) << 32) ^ \
    (((uint64_t) ((uint8_t *) (p))[5]) << 40) ^ \
    (((uint64_t) ((uint8_t *) (p))[6]) << 48) ^ \
    (((uint64_t) ((uint8_t *) (p))[7]) << 56))
#endif

// Cyclic right rotation
#ifndef ROTR64
#define ROTR64(x, y)  (((x) >> (y)) ^ ((x) << (64 - (y))))
#endif

// G mixing function
#ifndef B2B_G
#define B2B_G(a, b, c, d, x, y)     \
{                                   \
    v[a] = v[a] + v[b] + x;         \
    v[d] = ROTR64(v[d] ^ v[a], 32); \
    v[c] = v[c] + v[d];             \
    v[b] = ROTR64(v[b] ^ v[c], 24); \
    v[a] = v[a] + v[b] + y;         \
    v[d] = ROTR64(v[d] ^ v[a], 16); \
    v[c] = v[c] + v[d];             \
    v[b] = ROTR64(v[b] ^ v[c], 63); \
}
#endif

////////////////////////////////////////////////////////////////////////////////
//  Hash message
////////////////////////////////////////////////////////////////////////////////
void partialHash(
    // context
    blake2b_ctx * ctx,
    // optional secret key
    const void * key,
    // message
    const void * mes,
    // message length in bytes
    uint32_t meslen
) {
    const uint64_t blake2b_iv[8] = {
        0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
        0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
        0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
        0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
    };

    const uint8_t sigma[12][16] = {
        { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
        { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
        { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
        { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
        { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
        { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
        { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
        { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
        { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
        { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
        { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
        { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
    };

    int i;
    int j;

    uint64_t v[16];
    uint64_t m[16];

    //====================================================================//
    //  Initialize context
    //====================================================================//
    for (j = 0; j < 8; ++j)
    {
        ctx->h[j] = blake2b_iv[j];
    }

    ctx->h[0] ^= 0x01010000 ^ (KEY_LEN << 8) ^ HASH_LEN;

    ctx->t[0] = 0;
    ctx->t[1] = 0;
    ctx->c = 0;

    for (j = KEY_LEN; j < 128; ++j)
    {
        ctx->b[j] = 0;
    }

    //====================================================================//
    //  Hash key [optional]
    //====================================================================//
    for (j = 0; j < KEY_LEN; ++j)
    {
        if (ctx->c == 128)
        {
            ctx->t[0] += ctx->c;
            ctx->t[1] += (ctx->t[0] < ctx->c)? 1: 0;

            for (i = 0; i < 8; ++i)
            {
                v[i] = ctx->h[i];
                v[i + 8] = blake2b_iv[i];
            }

            v[12] ^= ctx->t[0];
            v[13] ^= ctx->t[1];

            for (i = 0; i < 16; i++)
            {
                m[i] = B2B_GET64(&ctx->b[8 * i]);
            }

            for (i = 0; i < 12; ++i)
            {
                B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
                B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
                B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
                B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
                B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
                B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
                B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
                B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
            }

            for (i = 0; i < 8; ++i)
            {
                ctx->h[i] ^= v[i] ^ v[i + 8];
            }

            ctx->c = 0;
        }

        ctx->b[ctx->c++] = ((const uint8_t *)key)[j];
    }

    if (KEY_LEN > 0)
    {
        ctx->c = 128;
    }

    //====================================================================//
    //  Hash message
    //====================================================================//
    for (j = 0; j < meslen; ++j)
    {
        if (ctx->c == 128)
        {
            ctx->t[0] += ctx->c;
            ctx->t[1] += (ctx->t[0] < ctx->c)? 1: 0;

            for (i = 0; i < 8; ++i)
            {
                v[i] = ctx->h[i];
                v[i + 8] = blake2b_iv[i];
            }

            v[12] ^= ctx->t[0];
            v[13] ^= ctx->t[1];

            for (i = 0; i < 16; i++)
            {
                m[i] = B2B_GET64(&ctx->b[8 * i]);
            }

            for (i = 0; i < 12; ++i)
            {
                B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
                B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
                B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
                B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
                B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
                B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
                B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
                B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
            }

            for (i = 0; i < 8; ++i)
            {
                ctx->h[i] ^= v[i] ^ v[i + 8];
            }

            ctx->c = 0;
        }

        ctx->b[ctx->c++] = ((const uint8_t *)mes)[j];
    }

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Block mining                                                               
////////////////////////////////////////////////////////////////////////////////
__global__ void blockMining(
    const uint32_t * data,
    // pregenerated nonces
    const void * non,
    // precalculated hashes
    const void * hash,
    // results
    uint32_t * res
) {
    const uint64_t blake2b_iv[8] = {
        0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
        0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
        0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
        0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
    };

    const uint8_t sigma[12][16] = {
        { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
        { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
        { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
        { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
        { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
        { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
        { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
        { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
        { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
        { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
        { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
        { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
    };

    // local
    uint64_t v[16];
    uint64_t m[16];
    uint32_t ind[K_SIZE];
    // 5 * 64 bits, if HASH_LEN == 32
    uint8_t h[HASH_LEN + 4];

    uint32_t j;
    uint32_t tid = threadIdx.x;
    __shared__ uint32_t shm[64];

    shm[tid] = data[tid];
    __syncthreads();

    // shared
    uint32_t * key = (uint32_t *)shm;
    blake2b_ctx * ctx;

#pragma unroll
    for (int l = 0; l < H_SIZE; ++l) 
    {
        ctx = (blake2b_ctx *)(shm + 8);

        tid = (
            threadIdx.x + blockDim.x * blockIdx.x + l * gridDim.x * blockDim.x
        ) << 3;

        const uint8_t * mes = (const uint8_t *)((const uint32_t *)non + tid);

    //====================================================================//
    //  Hash nonce
    //====================================================================//
        for (j = 0; ctx->c < 128 && j < NON_LEN; ++j)
        {
            ctx->b[ctx->c++] = mes[j];
        }

        while (j < NON_LEN)
        {
            ctx->t[0] += ctx->c;
            ctx->t[1] += 1 - !(ctx->t[0] < ctx->c);

#pragma unroll
            for (int i = 0; i < 8; ++i)
            {
                v[i] = ctx->h[i];
                v[i + 8] = blake2b_iv[i];
            }

            v[12] ^= ctx->t[0];
            v[13] ^= ctx->t[1];

#pragma unroll
            for (int i = 0; i < 16; i++)
            {
                m[i] = B2B_GET64(&ctx->b[8 * i]);
            }

#pragma unroll
            for (int i = 0; i < 12; ++i)
            {
                B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
                B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
                B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
                B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
                B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
                B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
                B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
                B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
            }

#pragma unroll
            for (int i = 0; i < 8; ++i)
            {
                ctx->h[i] ^= v[i] ^ v[i + 8];
            }

            ctx->c = 0;
           
            while (ctx->c < 128 && j < NON_LEN)
            {
                ctx->b[ctx->c++] = mes[j++];
            }
        }

    //====================================================================//
    //  Finalize h
    //====================================================================//
        ctx->t[0] += ctx->c;
        ctx->t[1] += 1 - !(ctx->t[0] < ctx->c);

        while (ctx->c < 128)
        {
            ctx->b[ctx->c++] = 0;
        }

#pragma unroll
        for (int i = 0; i < 8; ++i)
        {
            v[i] = ctx->h[i];
            v[i + 8] = blake2b_iv[i];
        }

        v[12] ^= ctx->t[0];
        v[13] ^= ctx->t[1];
        v[14] = ~v[14];

#pragma unroll
        for (int i = 0; i < 16; i++)
        {
            m[i] = B2B_GET64(&ctx->b[8 * i]);
        }

#pragma unroll
        for (int i = 0; i < 12; ++i)
        {
            B2B_G(0, 4,  8, 12, m[sigma[i][ 0]], m[sigma[i][ 1]]);
            B2B_G(1, 5,  9, 13, m[sigma[i][ 2]], m[sigma[i][ 3]]);
            B2B_G(2, 6, 10, 14, m[sigma[i][ 4]], m[sigma[i][ 5]]);
            B2B_G(3, 7, 11, 15, m[sigma[i][ 6]], m[sigma[i][ 7]]);
            B2B_G(0, 5, 10, 15, m[sigma[i][ 8]], m[sigma[i][ 9]]);
            B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
            B2B_G(2, 7,  8, 13, m[sigma[i][12]], m[sigma[i][13]]);
            B2B_G(3, 4,  9, 14, m[sigma[i][14]], m[sigma[i][15]]);
        }

#pragma unroll
        for (int i = 0; i < 8; ++i)
        {
            ctx->h[i] ^= v[i] ^ v[i + 8];
        }

        for (j = 0; j < HASH_LEN; ++j)
        {
            h[j] = (ctx->h[j >> 3] >> ((j & 7) << 3)) & 0xFF;
        }

    //===================================================================//
    //  Generate indices
    //===================================================================//
#pragma unroll
        for (int i = 0; i < 3; ++i)
        {
            h[HASH_LEN + i] = h[i];
        }

#pragma unroll
        for (int i = 0; i < K_SIZE; ++i)
        {
            ind[i] = *((uint32_t *)(h + i)) & 0x03FFFFFF;
        }
        
    //===================================================================//
    //  Calculate result
    //===================================================================//
        uint32_t * r = (uint32_t *)h;

        // first addition of hashes -> r
        asm volatile (
            "add.cc.u32 %0, %1, %2;":
            "=r"(h[0]): "r"(hash[ind[0]][0]), "r"(hash[ind[1]][0])
        );

#pragma unroll
        for (int i = 1; i < 8; ++i)
        {
            asm volatile (
                "addc.cc.u32 %0, %1, %2;":
                "=r"(h[i]): "r"(hash[ind[0]][i]), "r"(hash[ind[1]][i])
            );
        }

        asm volatile (
            "addc.u32 %0, 0, 0;": "=r"(r[8])
        );

        // remaining additions
#pragma unroll
        for (int k = 2; k < K_SIZE; ++k)
        {
            asm volatile (
                "add.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(hash[ind[k]][0])
            );

#pragma unroll
            for (int i = 1; i < 8; ++i)
            {
                asm volatile (
                    "addc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(hash[ind[k]][i])
                );
            }

            asm volatile (
                "addc.u32 %0, %0, 0;": "+r"(r[8])
            );
        }

        // subtraction of secret key
        asm volatile (
            "sub.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(key[0])
        );

#pragma unroll
        for (int i = 1; i < 8; ++i)
        {
            asm volatile (
                "subc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(key[i])
            );
        }

        asm volatile (
            "subc.u32 %0, %0, 0;": "+r"(r[8])
        );


    //===================================================================//
    //  result mod q
    //===================================================================//
        uint32_t * med = ind;
        uint32_t * d = ind + 5; 
        uint32_t * carry = ind + 6;

        *d = (r[8] << 4) | (r[7] >> 28);

        // correct highest 32 bits
        r[7] &= 0x0FFFFFFF;

    //====================================================================//
        asm volatile (
            "mul.lo.u32 %0, %1, "q0_s";": "=r"(med[0]): "r"(*d)
        );
        asm volatile (
            "mul.hi.u32 %0, %1, "q0_s";": "=r"(med[1]): "r"(*d)
        );
        asm volatile (
            "mul.lo.u32 %0, %1, "q2_s";": "=r"(med[2]): "r"(*d)
        );
        asm volatile (
            "mul.hi.u32 %0, %1, "q2_s";": "=r"(med[3]): "r"(*d)
        );

        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q1_s", %0;": "+r"(med[1]): "r"(*d)
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q1_s", %0;": "+r"(med[2]): "r"(*d)
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_s", %0;": "+r"(med[3]): "r"(*d)
        );
        asm volatile (
            "madc.hi.u32 %0, %1, "q3_s", 0;": "=r"(med[4]): "r"(*d)
        );

    //====================================================================//
        asm volatile (
            "sub.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(med[0])
        );

#pragma unroll
        for (int i = 1; i < 5; ++i)
        {
            asm volatile (
                "subc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(med[i])
            );
        }

#pragma unroll
        for (int i = 5; i < 8; ++i)
        {
            asm volatile (
                "subc.cc.u32 %0, %0, 0;": "+r"(r[i])
            );
        }

    //====================================================================//
        asm volatile (
            "subc.u32 %0, 0, 0;": "=r"(*carry)
        );

        *carry = 0 - *carry;

        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q0_s", %0;": "+r"(r[0]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q1_s", %0;": "+r"(r[1]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q2_s", %0;": "+r"(r[2]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_s", %0;": "+r"(r[3]): "r"(*carry)
        );

#pragma unroll
        for (int i = 0; i < 3; ++i)
        {
            asm volatile (
                "addc.cc.u32 %0, %0, 0;": "+r"(r[i + 4])
            );
        }

        asm volatile (
            "addc.u32 %0, %0, 0;": "+r"(r[7])
        );

    //===================================================================//
    //  dump result to global memory
    //===================================================================//
#pragma unroll
        for (int i = 0; i < 3; ++i)
        {
            res[tid + i] = r[i];
        }
    }

    return;
}
