#include "hip/hip_runtime.h"
// mining.cu

/*******************************************************************************

    MINING -- Autolykos parallel BlockMining procedure

*******************************************************************************/

#include "../include/mining.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//  Unfinalized hash of message
////////////////////////////////////////////////////////////////////////////////
void InitMining(
    // context
    blake2b_ctx * ctx,
    // message
    const uint32_t * mes,
    // message length in bytes
    const uint32_t meslen
)
{
    int j;

    uint64_t aux[32];

    //====================================================================//
    //  Initialize context
    //====================================================================//
    B2B_IV(ctx->h);

    ctx->h[0] ^= 0x01010000 ^ (0 << 8) ^ NUM_SIZE_8;
    ctx->t[0] = 0;
    ctx->t[1] = 0;
    ctx->c = 0;

    for (j = 0; j < 128; ++j)
    {
        ctx->b[j] = 0;
    }

    //====================================================================//
    //  Hash message
    //====================================================================//
    for (j = 0; j < meslen; ++j)
    {
        if (ctx->c == 128)
        {
            B2B_H_HOST(ctx, aux);
        }

        ctx->b[ctx->c++] = ((const uint8_t *)mes)[j];
    }

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Block mining                                                               
////////////////////////////////////////////////////////////////////////////////
__global__ void BlockMining(
    // boundary for puzzle
    const uint32_t * bound,
    // data: pk || mes || w || padding || x || sk || ctx
    const uint32_t * data,
    // pregenerated nonces
    const uint32_t * non,
    // precalculated hashes
    const uint32_t * hash,
    // results
    uint32_t * res,
    // indices of valid solutions
    uint32_t * valid
)
{
    uint32_t j;
    uint32_t tid = threadIdx.x;

    // shared memory
    // B_DIM * 4 bytes  
    __shared__ uint32_t sdata[B_DIM];

    // B_DIM * 4 bytes
    sdata[tid] = data[tid + PK2_SIZE_32 + 2 * NUM_SIZE_32];
    __syncthreads();

    // NUM_SIZE_8 bytes
    uint32_t * sk = sdata;

    // local memory
    // 472 bytes
    uint32_t ldata[118];

    // 256 bytes
    uint64_t * aux = (uint64_t *)ldata;
    // (4 * K_LEN) bytes
    uint32_t * ind = ldata;
    // (NUM_SIZE_8 + 4) bytes
    uint32_t * r = ind + K_LEN;
    // (212 + 4) bytes 
    blake2b_ctx * ctx = (blake2b_ctx *)(ldata + 64);

#pragma unroll
    for (int l = 0; l < H_LEN; ++l) 
    {
        *ctx = *((blake2b_ctx *)(sdata + NUM_SIZE_32));

        tid = threadIdx.x + blockDim.x * blockIdx.x
            + l * gridDim.x * blockDim.x;

        const uint8_t * mes = (const uint8_t *)(non + tid * NONCE_SIZE_32);

    //====================================================================//
    //  Hash nonce
    //====================================================================//
#pragma unroll
        for (j = 0; ctx->c < 128 && j < NONCE_SIZE_8; ++j)
        {
            ctx->b[ctx->c++] = mes[j];
        }

#pragma unroll
        for ( ; j < NONCE_SIZE_8; )
        {
            B2B_H(ctx, aux);
           
#pragma unroll
            for ( ; ctx->c < 128 && j < NONCE_SIZE_8; ++j)
            {
                ctx->b[ctx->c++] = mes[j];
            }
        }

    //====================================================================//
    //  Finalize hash
    //====================================================================//
        B2B_H_LAST(ctx, aux);

#pragma unroll
        for (j = 0; j < NUM_SIZE_8; ++j)
        {
            ((uint8_t *)r)[(j & 0xFFFFFFFC) + (3 - (j & 3))]
                = (ctx->h[j >> 3] >> ((j & 7) << 3)) & 0xFF;
        }

    //===================================================================//
    //  Generate indices
    //===================================================================//
#pragma unroll
        for (int i = 1; i < 4; ++i)
        {
            ((uint8_t *)r)[NUM_SIZE_8 + i] = ((uint8_t *)r)[i];
        }

#pragma unroll
        for (int k = 0; k < K_LEN; k += 4) 
        { 
            ind[k] = r[k >> 2] & N_MASK; 
        
#pragma unroll 
            for (int i = 1; i < 4; ++i) 
            { 
                ind[k + i] 
                    = (
                        (r[k >> 2] << (i << 3))
                        | (r[(k >> 2) + 1] >> (32 - (i << 3)))
                    ) & N_MASK; 
            } 
        } 

    //===================================================================//
    //  Calculate result
    //===================================================================//
        // first addition of hashes -> r
        asm volatile (
            "add.cc.u32 %0, %1, %2;":
            "=r"(r[0]): "r"(hash[ind[0] << 3]), "r"(hash[ind[1] << 3])
        );

#pragma unroll
        for (int i = 1; i < 8; ++i)
        {
            asm volatile (
                "addc.cc.u32 %0, %1, %2;":
                "=r"(r[i]):
                "r"(hash[(ind[0] << 3) + i]), "r"(hash[(ind[1] << 3) + i])
            );
        }

        asm volatile ("addc.u32 %0, 0, 0;": "=r"(r[8]));

     // remaining additions
#pragma unroll
        for (int k = 2; k < K_LEN; ++k)
        {
            asm volatile (
                "add.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(hash[ind[k] << 3])
            );

#pragma unroll
            for (int i = 1; i < 8; ++i)
            {
                asm volatile (
                    "addc.cc.u32 %0, %0, %1;":
                    "+r"(r[i]): "r"(hash[(ind[k] << 3) + i])
                );
            }

            asm volatile ("addc.u32 %0, %0, 0;": "+r"(r[8]));
        }

        // subtraction of secret key
        asm volatile ("sub.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(sk[0]));

#pragma unroll
        for (int i = 1; i < 8; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(sk[i]));
        }

        asm volatile ("subc.u32 %0, %0, 0;": "+r"(r[8]));

    //===================================================================//
    //  Result mod Q
    //===================================================================//
        // 20 bytes
        uint32_t * med = ind;
        // 4 bytes
        uint32_t * d = ind + 5; 
        uint32_t * carry = d;

        d[0] = r[8];

    //====================================================================//
        asm volatile ("mul.lo.u32 %0, %1, "q0_s";": "=r"(med[0]): "r"(*d));
        asm volatile ("mul.hi.u32 %0, %1, "q0_s";": "=r"(med[1]): "r"(*d));
        asm volatile ("mul.lo.u32 %0, %1, "q2_s";": "=r"(med[2]): "r"(*d));
        asm volatile ("mul.hi.u32 %0, %1, "q2_s";": "=r"(med[3]): "r"(*d));

        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q1_s", %0;": "+r"(med[1]): "r"(*d)
        );

        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q1_s", %0;": "+r"(med[2]): "r"(*d)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_s", %0;": "+r"(med[3]): "r"(*d)
        );

        asm volatile ("madc.hi.u32 %0, %1, "q3_s", 0;": "=r"(med[4]): "r"(*d));

    //====================================================================//
        asm volatile ("sub.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(med[0]));

#pragma unroll
        for (int i = 1; i < 5; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(med[i]));
        }

#pragma unroll
        for (int i = 5; i < 7; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, 0;": "+r"(r[i]));
        }

        asm volatile ("subc.u32 %0, %0, 0;": "+r"(r[7]));

    //====================================================================//
        d[1] = d[0] >> 31;
        d[0] <<= 1;

        asm volatile ("add.cc.u32 %0, %0, %1;": "+r"(r[4]): "r"(d[0]));
        asm volatile ("addc.cc.u32 %0, %0, %1;": "+r"(r[5]): "r"(d[1]));
        asm volatile ("addc.cc.u32 %0, %0, 0;": "+r"(r[6]));
        asm volatile ("addc.u32 %0, %0, 0;": "+r"(r[7]));

    //====================================================================//
        asm volatile ("sub.cc.u32 %0, %0, "q0_s";": "+r"(r[0]));
        asm volatile ("subc.cc.u32 %0, %0, "q1_s";": "+r"(r[1]));
        asm volatile ("subc.cc.u32 %0, %0, "q2_s";": "+r"(r[2]));
        asm volatile ("subc.cc.u32 %0, %0, "q3_s";": "+r"(r[3]));
        asm volatile ("subc.cc.u32 %0, %0, "q4_s";": "+r"(r[4]));

#pragma unroll
        for (int i = 5; i < 8; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, "qhi_s";": "+r"(r[i]));
        }

        asm volatile ("subc.u32 %0, 0, 0;": "=r"(*carry));

        *carry = 0 - *carry;

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q0_s", %0;": "+r"(r[0]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q1_s", %0;": "+r"(r[1]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q2_s", %0;": "+r"(r[2]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_s", %0;": "+r"(r[3]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q4_s", %0;": "+r"(r[4]): "r"(*carry)
        );

#pragma unroll
        for (int i = 5; i < 7; ++i)
        {
            asm volatile (
                "madc.lo.cc.u32 %0, %1, "qhi_s", %0;": "+r"(r[i]): "r"(*carry)
            );
        }

        asm volatile (
            "madc.lo.u32 %0, %1, "qhi_s", %0;": "+r"(r[7]): "r"(*carry)
        );

    //===================================================================//
    //  Dump result to global memory -- LITTLE ENDIAN
    //===================================================================//
        j = ((uint64_t *)r)[3] < ((uint64_t *)bound)[3]
            || ((uint64_t *)r)[3] == ((uint64_t *)bound)[3] && (
                ((uint64_t *)r)[2] < ((uint64_t *)bound)[2]
                || ((uint64_t *)r)[2] == ((uint64_t *)bound)[2] && (
                    ((uint64_t *)r)[1] < ((uint64_t *)bound)[1]
                    || ((uint64_t *)r)[1] == ((uint64_t *)bound)[1]
                    && ((uint64_t *)r)[0] <= ((uint64_t *)bound)[0]
                )
            );

        valid[tid] = (1 - !j) * (tid + 1);

#pragma unroll
        for (int i = 0; i < NUM_SIZE_32; ++i)
        {
            res[tid * NUM_SIZE_32 + i] = r[i];
        }

        __syncthreads();
    }

    return;
}

// mining.cu
