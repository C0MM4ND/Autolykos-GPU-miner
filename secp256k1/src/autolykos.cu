#include "hip/hip_runtime.h"
// autolykos.cu

/*******************************************************************************

    AUTOLYKOS -- Autolykos puzzle cycle

*******************************************************************************/

#include "../include/compaction.h"
#include "../include/conversion.h"
#include "../include/mining.h"
#include "../include/prehash.h"
#include "../include/request.h"
#include "../include/reduction.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <sys/types.h>
#include <inttypes.h>
#include <unistd.h>
#include <curl/curl.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <openssl/bn.h>
#include <openssl/bio.h>
#include <openssl/err.h>
#include <openssl/ec.h>
#include <openssl/pem.h>

////////////////////////////////////////////////////////////////////////////////
//  Read secret key
////////////////////////////////////////////////////////////////////////////////
int ReadSecKey(
    char * filename,
    void * sk
)
{
    FILE * in = fopen(filename, "r");

    int status;

    for (int i = 0; i < NUM_SIZE_64; ++i)
    {
        status = fscanf(
            in, "%"SCNx64"\n", (uint64_t *)sk + NUM_SIZE_64 - i - 1
        );
    }

    fclose(in);

    return status;
}

////////////////////////////////////////////////////////////////////////////////
//  Generate key pair
////////////////////////////////////////////////////////////////////////////////
int GenerateKeyPair(
    uint8_t * sk,
    uint8_t * pk
)
{
    BIO * outbio = NULL;
    EC_KEY * eck = NULL;
    EVP_PKEY * evpk = NULL;
    int eccgrp;

    // initialize openssl
    OpenSSL_add_all_algorithms();
    ERR_load_BIO_strings();
    ERR_load_crypto_strings();

    // create Input/Output BIO's
    outbio = BIO_new(BIO_s_file());
    outbio = BIO_new_fp(stdout, BIO_NOCLOSE);

    // create EC key sructure
    // set group type from NID
    eccgrp = OBJ_txt2nid("secp256k1");
    eck = EC_KEY_new_by_curve_name(eccgrp);

    // OPENSSL_EC_NAMED_CURVE flag for cert signing
    EC_KEY_set_asn1_flag(eck, OPENSSL_EC_NAMED_CURVE);

    // create public/private EC key pair
    if (!(EC_KEY_generate_key(eck)))
    {
        BIO_printf(outbio, "Error generating the ECC key.");
    }

    // convert EC key into PKEY structure
    evpk = EVP_PKEY_new();
    if (!EVP_PKEY_assign_EC_KEY(evpk, eck))
    {
        BIO_printf(outbio, "Error assigning ECC key to EVP_PKEY structure.");
    }

    // Now we show how to extract EC-specifics from the key
    eck = EVP_PKEY_get1_EC_KEY(evpk);

    const EC_GROUP * ecgrp = EC_KEY_get0_group(eck);

    //====================================================================//
    //  Public key extraction
    //====================================================================//
    const EC_POINT * ecp = EC_KEY_get0_public_key(eck);

    char * str = EC_POINT_point2hex(
        ecgrp, ecp, POINT_CONVERSION_COMPRESSED, NULL
    );

    int len = 0;

    if (str)
    {
        for ( ; str[len] != '\0'; ++len) {}
    }
    else
    {
        printf("ERROR\n");
        fflush(stdout);
    }

    HexStrToBigEndian(str, len, pk, PK_SIZE_8);

    OPENSSL_free(str);
    str = NULL;

    //====================================================================//
    //  Secret key extraction
    //====================================================================//
    const BIGNUM * bn = EC_KEY_get0_private_key(eck);

    str = BN_bn2hex(bn);
    len = 0;

    if (str)
    {
        for ( ; str[len] != '\0'; ++len) {}
    }
    else
    {
        printf("ERROR\n");
        fflush(stdout);
    }

    HexStrToLittleEndian(str, len, sk, NUM_SIZE_8);

    OPENSSL_free(str);

    //====================================================================//
    //  Deallocation
    //====================================================================//
    EVP_PKEY_free(evpk);
    EC_KEY_free(eck);
    BIO_free_all(outbio);

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
//  Generate consequtive nonces
////////////////////////////////////////////////////////////////////////////////
__global__ void GenerateConseqNonces(
    uint64_t * arr,
    uint32_t len,
    uint64_t base
)
{
    uint32_t tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint64_t nonce = base + tid;

    INPLACE_REVERSE_ENDIAN(&nonce);

    if (tid < len) arr[tid] = nonce;

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
int main(
    int argc, char ** argv
)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (!deviceCount)
    {
        printf("ABORT: GPU devices did not recognised.");

        return 1;
    }

    curl_global_init(CURL_GLOBAL_ALL);

    //====================================================================//
    //  Host memory allocation
    //====================================================================//
    uint8_t state = 1;
    uint32_t ind = 0;
    uint64_t base = 0;

    string block;
    InitString(&block);

    // hash context
    // (212 + 4) bytes
    blake2b_ctx ctx_h;

    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t sk_h[NUM_SIZE_8];
    uint8_t pk_h[PK_SIZE_8];
    uint8_t x_h[NUM_SIZE_8];
    uint8_t w_h[PK_SIZE_8];
    uint8_t res_h[NUM_SIZE_8];
    uint8_t nonce_h[NONCE_SIZE_8];

    char filename[10] = "./seckey";

    if (argc == 1)
    {
        printf("Use secret key from './seckey'\n");
        fflush(stdout);

        if (access(filename, F_OK) == -1)
        {
            printf("ABORT: File \"./seckey\" not found\n");

            return 1;
        }
    }
    else
    {
        if (access(argv[1], F_OK) == -1)
        {
            printf("ABORT: File not found\n");

            return 1;
        }
    }

    ReadSecKey((argc == 1)? filename: argv[1], sk_h);

    //====================================================================//
    //  Device memory allocation
    //====================================================================//
    // boundary for puzzle
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc((void **)&bound_d, NUM_SIZE_8));

    // nonces
    // H_LEN * L_LEN * NONCE_SIZE_8 bytes // 32 MB
    uint32_t * nonce_d;
    CUDA_CALL(hipMalloc((void **)&nonce_d, H_LEN * L_LEN * NONCE_SIZE_8));

    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, (NUM_SIZE_8 + B_DIM) * 4));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GB
    uint32_t * hash_d;
    CUDA_CALL(hipMalloc((void **)&hash_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (H_LEN * N_LEN * 8 + 4) bytes // ~512 MB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc((void **)&indices_d, H_LEN * N_LEN * 8 + 4));

    // potential solutions of puzzle
    // H_LEN * L_LEN * 4 * 8 bytes // 16 * 8 MB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, H_LEN * L_LEN * NUM_SIZE_8));

    //====================================================================//
    //  Random generator initialization
    //====================================================================//
    /// original /// hiprandGenerator_t gen;
    /// original /// CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
    /// original ///
    /// original /// time_t rawtime;
    /// original /// // get current time (ms)
    /// original /// time(&rawtime);

    /// original /// // set seed
    /// original /// CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (uint64_t)rawtime));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    while (1)
    {
        GetLatestBlock(&block, bound_h, mes_h, pk_h, &state);

        // state is changed
        if (state)
        {
            // copy boundary
            CUDA_CALL(hipMemcpy(
                (void *)bound_d, (void *)bound_h, NUM_SIZE_8,
                hipMemcpyHostToDevice
            ));

            // copy public key
            CUDA_CALL(hipMemcpy(
                (void *)data_d, (void *)pk_h, PK_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy message
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8), (void *)mes_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy secret key
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + PK2_SIZE_32 + 2 * NUM_SIZE_32), (void *)sk_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // generate one-time key pair
            GenerateKeyPair(x_h, w_h);

            // copy one time secret key
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + PK2_SIZE_32 + NUM_SIZE_32), (void *)x_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time public key
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8),
                (void *)w_h, PK_SIZE_8, hipMemcpyHostToDevice
            ));

            // precalculate hashes
            Prehash(data_d, hash_d, indices_d);

            state = 0;

            printf("Prehash finished\n");
            fflush(stdout);

            printf(
                "m     = 0x%016lX %016lX %016lX %016lX\n",
                ((uint64_t *)mes_h)[3], ((uint64_t *)mes_h)[2],
                ((uint64_t *)mes_h)[1], ((uint64_t *)mes_h)[0]
            );

            printf(
                "pk    = 0x%02lX %016lX %016lX %016lX %016lX\n",
                ((uint8_t *)pk_h)[0],
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 0),
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 1),
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 2),
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)pk_h + 1)) + 3)
            );

            printf(
                "sk     = 0x%016lX %016lX %016lX %016lX\n",
                ((uint64_t *)sk_h)[3], ((uint64_t *)sk_h)[2],
                ((uint64_t *)sk_h)[1], ((uint64_t *)sk_h)[0]
            );

            printf(
                "w     = 0x%02lX %016lX %016lX %016lX %016lX\n",
                ((uint8_t *)w_h)[0],
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 0),
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 1),
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 2),
                REVERSE_ENDIAN(((uint64_t *)((uint8_t *)w_h + 1)) + 3)
            );

            printf(
                "x     = 0x%016lX %016lX %016lX %016lX\n",
                ((uint64_t *)x_h)[3], ((uint64_t *)x_h)[2],
                ((uint64_t *)x_h)[1], ((uint64_t *)x_h)[0]
            );

            printf(
                "b     = 0x%016lX %016lX %016lX %016lX\n",
                ((uint64_t *)bound_h)[3],
                ((uint64_t *)bound_h)[2],
                ((uint64_t *)bound_h)[1],
                ((uint64_t *)bound_h)[0]
            );
        }

        CUDA_CALL(hipDeviceSynchronize());

        // generate nonces
        /// original /// CURAND_CALL(hiprandGenerate(gen, nonce_d, H_LEN * L_LEN * NONCE_SIZE_8));
        GenerateConseqNonces<<<1 + (H_LEN * L_LEN - 1) / B_DIM, B_DIM>>>(
            (uint64_t *)nonce_d, N_LEN, base
        );
        base += H_LEN * L_LEN;

        // calculate unfinalized hash of message
        InitMining(&ctx_h, (uint32_t *)mes_h, NUM_SIZE_8);

        // copy context
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + PK2_SIZE_32 + 3 * NUM_SIZE_32), (void *)&ctx_h,
            sizeof(blake2b_ctx), hipMemcpyHostToDevice
        ));

        // calculate solution candidates
        BlockMining<<<1 + (L_LEN - 1) / B_DIM, B_DIM>>>(
            bound_d, data_d, nonce_d, hash_d, res_d, indices_d
        );

        // try to find solution
        ind = FindNonZero(indices_d, indices_d + H_LEN * L_LEN, H_LEN * L_LEN);

        if (ind)
        {
            CUDA_CALL(hipMemcpy(
                (void *)res_h, (void *)(res_d + ((ind - 1) << 3)), NUM_SIZE_8,
                hipMemcpyDeviceToHost
            ));

            CUDA_CALL(hipMemcpy(
                (void *)nonce_h, (void *)(nonce_d + ((ind - 1) << 1)),
                NONCE_SIZE_8, hipMemcpyDeviceToHost
            ));

            // printf("TRY");
            // fflush(stdout);

            printf(
                "nonce = 0x%016lX\n",
                ((uint64_t *)nonce_h)[0]
            );

            printf(
                "d     = 0x%016lX %016lX %016lX %016lX\n",
                ((uint64_t *)res_h)[3],
                ((uint64_t *)res_h)[2],
                ((uint64_t *)res_h)[1],
                ((uint64_t *)res_h)[0]
            );

            PostPuzzleSolution(w_h, nonce_h, res_h);

            state = 1;
        }

        struct timeval tmo;
        fd_set readfds;

        //printf(".");
        //fflush(stdout);

        FD_ZERO(&readfds);
        FD_SET(0, &readfds);
        tmo.tv_sec = 0.0001;
        tmo.tv_usec = 0;

        switch (select(1, &readfds, NULL, NULL, &tmo))
        {
            case -1:
                printf("Commencing termination\n");
                fflush(stdout);
                break;
            case 0:
                continue;
        }

        if (getchar() == 'e') {
            printf("Commencing termination\n");
            fflush(stdout);
            break;
        }
    }

    hipDeviceSynchronize();

    //====================================================================//
    //  Free device memory
    //====================================================================//
    /// original /// CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(bound_d));
    CUDA_CALL(hipFree(nonce_d));
    CUDA_CALL(hipFree(hash_d));
    CUDA_CALL(hipFree(data_d));
    CUDA_CALL(hipFree(indices_d));
    CUDA_CALL(hipFree(res_d));

    //====================================================================//
    //  Free host memory
    //====================================================================//

    if (block.ptr)
    {
        free(block.ptr);
    }

    curl_global_cleanup();

    return 0;
}

// autolykos.cu
