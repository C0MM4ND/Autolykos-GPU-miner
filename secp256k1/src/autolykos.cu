// autolykos.cu

/*******************************************************************************

    AUTOLYKOS -- Autolykos puzzle cycle

*******************************************************************************/

#include "../include/compaction.h"
#include "../include/conversion.h"
#include "../include/cryptography.h"
#include "../include/definitions.h"
#include "../include/jsmn.h"
#include "../include/mining.h"
#include "../include/prehash.h"
#include "../include/processing.h"
#include "../include/reduction.h"
#include "../include/request.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <curl/curl.h>
#include <inttypes.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>
#include <atomic>
#include <thread>
#include <chrono>
#include <mutex>
#include <vector>
#include <iostream>
#define TEXT_SEPARATOR   "========================================"\
                         "========================================\n"
#define TEXT_GPUCHECK    " Checking GPU availability\n"
#define TEXT_TERMINATION " Miner is now terminated\n"
#define ERROR_GPUCHECK   "ABORT:  GPU devices are not recognised\n"

using namespace std::chrono;


struct globalInfo
{
    // Puzzle data to read
    
    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t sk_h[NUM_SIZE_8];
    uint8_t pk_h[PK_SIZE_8];
    char skstr[NUM_SIZE_4];
    char pkstr[PK_SIZE_4 + 1];
    int keepPrehash;
    char to[40];
    // Mutex for reading/writing data from globalInfo safely

    std::mutex info_mutex;
    
    // Mutex for curl usage/maybe future websocket
    //not used now

    std::mutex io_mutex;

    // Increment when new block is sent by node

    std::atomic<unsigned int> blockId; 
};

void minerThread(int deviceId, globalInfo *info);


int main(int argc, char* argv[])
{
    int deviceCount;
    timestamp_t stamp;
    int status = EXIT_SUCCESS;
    globalInfo info;
    info.blockId = 1;
    state_t state = STATE_CONTINUE;
    if (hipGetDeviceCount(&deviceCount) != hipSuccess)
    {
        fprintf(
            stderr, ERROR_GPUCHECK "%s" TEXT_TERMINATION TEXT_SEPARATOR,
            TimeStamp(&stamp)
        );

        return EXIT_FAILURE;
    }

    printf("Using %i CUDA devices\n",deviceCount);

    PERSISTENT_CALL_STATUS(curl_global_init(CURL_GLOBAL_ALL), CURLE_OK);
	

    char confname[14] = "./config.json";
    char * filename = (argc == 1)? confname: argv[1];
    char from[40];
    char to[40];
    int diff;
   // int keepPrehash = 0;
    json_t request(0, REQ_LEN);
    
    printf(
        "Using configuration from \'%s\'\n", filename
    );
    fflush(stdout);

    // check access to config file
    if (access(filename, F_OK) == -1)
    {
        fprintf(stderr, "ABORT:  File \'%s\' not found\n", filename);

        fprintf(
            stderr, "%s" TEXT_TERMINATION TEXT_SEPARATOR, TimeStamp(&stamp)
        );

        return EXIT_FAILURE;
    }

    // read config from file
    status = ReadConfig(filename, info.sk_h, info.skstr, from, info.to, &info.keepPrehash, &stamp);

    if (status == EXIT_FAILURE)
    {
        fprintf(stderr, "ABORT:  Wrong config format\n");

        fprintf(
            stderr, "%s" TEXT_TERMINATION TEXT_SEPARATOR, TimeStamp(&stamp)
        );

        return EXIT_FAILURE;
    }

    // generate public key from secret key
    GeneratePublicKey(info.skstr, info.pkstr, info.pk_h);

    printf(
        "%s Generated public key:\n"
        "   pk = 0x%02lX %016lX %016lX %016lX %016lX\n",
        TimeStamp(&stamp), ((uint8_t *)info.pk_h)[0],
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 0),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 1),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 2),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 3)
    );
    fflush(stdout);
    

    status = GetLatestBlock(
        from, info.pkstr, &request, info.bound_h, info.mes_h, &state, &diff
    );
    
    std::vector<std::thread> miners(deviceCount);
    for(int i = 0; i < deviceCount; i++)
    {
        miners[i] = std::thread(minerThread, i, &info);

    }

    // main cycle - bomb node with HTTP with 10ms intervals, if new block came 
    //-> signal miners with blockId
    int curlcnt = 0;
    const int curltimes = 2000;
    //time_t differ = 0;

    //using namespace std::chrono;
    milliseconds ms = milliseconds::zero(); 

    while(!TerminationRequestHandler())
    {
        milliseconds start = duration_cast< milliseconds >(
            system_clock::now().time_since_epoch()
            );
        info.info_mutex.lock();
        // need to fix state somehow
        state = STATE_CONTINUE;
        
        status = GetLatestBlock(
            from, info.pkstr, &request, info.bound_h, info.mes_h, &state, &diff
        );
        
        if(status != EXIT_SUCCESS)
	    {
	        printf("Getting block error\n");
	    }
        info.info_mutex.unlock();

        ms +=  duration_cast< milliseconds >(system_clock::now().time_since_epoch()) - start;
        curlcnt++;
        if(curlcnt%curltimes == 0)
        {
            //printf("Average curling time %lf\n",(double)differ/(CLOCKS_PER_SEC*curltimes));
            std::cout << "Average curling time " << ms.count()/(double)curltimes << " ms \n";
            ms = milliseconds::zero();
        }

        if(diff || state == STATE_REHASH)
        {
            info.blockId++;
            diff = 0;
            printf("Got new block in main thread\n");
	        fflush(stdout);
	    }
        std::this_thread::sleep_for(std::chrono::milliseconds(10));

    }    


    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
void minerThread(int deviceId, globalInfo *info)
{
    int status = EXIT_SUCCESS;
    timestamp_t stamp;
    state_t state = STATE_KEYGEN;
    hipSetDevice(deviceId);
    
    //====================================================================//
    //  Host memory allocation
    //====================================================================//
    // curl http request
    json_t request(0, REQ_LEN);

    // hash context
    // (212 + 4) bytes
    context_t ctx_h;

    // autolykos variables
    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t sk_h[NUM_SIZE_8];
    uint8_t pk_h[PK_SIZE_8];
    uint8_t x_h[NUM_SIZE_8];
    uint8_t w_h[PK_SIZE_8];
    uint8_t res_h[NUM_SIZE_8];
    uint8_t nonces_h[NONCE_SIZE_8];

    // cryptography variables
    char skstr[NUM_SIZE_4];
    char pkstr[PK_SIZE_4 + 1];
    char from[40];
    char to[40];
    int keepPrehash = 0;
    unsigned int blockId = 0;
    milliseconds start;	
    
    // Copy from global to thread local data
    //===============================================

    info->info_mutex.lock();

    memcpy(sk_h,info->sk_h, NUM_SIZE_8*sizeof(uint8_t));
    memcpy(mes_h, info->mes_h, NUM_SIZE_8*sizeof(uint8_t));
    memcpy(bound_h, info->bound_h, NUM_SIZE_8*sizeof(uint8_t));
    memcpy(pk_h, info->pk_h, PK_SIZE_8*sizeof(uint8_t));
    memcpy(pkstr, info->pkstr, (PK_SIZE_4+1)*sizeof(uint8_t));
    memcpy(skstr, info->skstr, NUM_SIZE_4*sizeof(uint8_t));
    memcpy(to, info->to, 40*sizeof(char));
   // blockId = info->blockId.load();
    keepPrehash = info->keepPrehash;
    
    info->info_mutex.unlock();
    
    //end copy
    //===============================


    //====================================================================//
    //  Device memory allocation
    //====================================================================//
    printf(" %s thread GPU %i allocating GPU memory\n", TimeStamp(&stamp), deviceId);
   // fflush(stdout);

    // boundary for puzzle
    // ~0 MiB
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc((void **)&bound_d, NUM_SIZE_8));

    // nonces
    // THREAD_LEN * LOAD_LEN * NONCE_SIZE_8 bytes // 32 MiB
    uint32_t * nonces_d;
    CUDA_CALL(hipMalloc(
        (void **)&nonces_d, THREAD_LEN * LOAD_LEN * NONCE_SIZE_8
    ));

    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MiB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, (NUM_SIZE_8 + BLOCK_DIM) * 4));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GiB
    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc((void **)&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (THREAD_LEN * N_LEN * 2 + 1) * INDEX_SIZE_8 bytes // ~512 MiB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc(
        (void **)&indices_d, (THREAD_LEN * N_LEN * 2 + 1) * INDEX_SIZE_8
    ));

    // potential solutions of puzzle
    // THREAD_LEN * LOAD_LEN * NUM_SIZE_8 bytes // 128 MiB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, THREAD_LEN * LOAD_LEN * NUM_SIZE_8));

    // unfinalized hash contexts
    // N_LEN * 80 bytes // 5 GiB
    ucontext_t * uctxs_d;

    if (keepPrehash)
    {
        CUDA_CALL(hipMalloc(
            (void **)&uctxs_d, (uint32_t)N_LEN * sizeof(ucontext_t)
        ));
    }

    //====================================================================//
    //  Key-pair transfer form host to device
    //====================================================================//
    // copy public key
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)pk_h, PK_SIZE_8, hipMemcpyHostToDevice
    ));

    // copy secret key
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + PK2_SIZE_32 + 2 * NUM_SIZE_32), (void *)sk_h,
        NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    //state_t state = STATE_KEYGEN;
    int diff = 0;
    uint32_t ind = 0;
    uint64_t base = 0;

    if (keepPrehash)
    {
        printf(
            "%s Preparing unfinalized hashes\n" TEXT_SEPARATOR,
            TimeStamp(&stamp)
        );
        fflush(stdout);


        UncompleteInitPrehash<<<1 + (N_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            data_d, uctxs_d
        );

        CUDA_CALL(hipDeviceSynchronize());
    }

    int cntCycles = 0;
    int NCycles = 100;
    start = duration_cast<milliseconds> (system_clock::now().time_since_epoch());
    do
    {
        
	    cntCycles++;
	    if(cntCycles%NCycles == 0)
	    {
            milliseconds timediff = duration_cast<milliseconds> (system_clock::now().time_since_epoch()) - start;
            printf("%lf MHashes per second on GPU %i \n", (double)LOAD_LEN*NCycles/((double)1000*timediff.count()), deviceId);
	        start = duration_cast<milliseconds> (system_clock::now().time_since_epoch());
	    }
	
        // if solution was found by this thread, wait for new block to come 
	    if(state == STATE_KEYGEN)
	    {
		    while(info->blockId.load() == blockId)
		    {}
		    state = STATE_CONTINUE;
	    }

	    unsigned int controlId = info->blockId.load();
        if(blockId != controlId)
        {
            //if info->blockId changed, read new message and bound to thread-local mem

            info->info_mutex.lock();
            memcpy(mes_h, info->mes_h, NUM_SIZE_8*sizeof(uint8_t));
            memcpy(bound_h, info->bound_h, NUM_SIZE_8*sizeof(uint8_t));
            /*
            for(int i = 0; i < NUM_SIZE_8; i++)
            {
                mes_h[i] = info->mes_h[i];
                bound_h[i] = info->bound_h[i];
            }
            */
            info->info_mutex.unlock();
            state = STATE_REHASH;
	        printf("Thread read new block data, blockid %i old %i\n",blockId,controlId);
	        blockId = controlId;
            
            GenerateKeyPair(x_h, w_h);
        
            //PrintPuzzleState(mes_h, pk_h, sk_h, w_h, x_h, bound_h, &stamp);

            // copy boundary
            CUDA_CALL(hipMemcpy(
                (void *)bound_d, (void *)bound_h, NUM_SIZE_8,
                hipMemcpyHostToDevice
            ));

            // copy message
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8), (void *)mes_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time secret key
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + PK2_SIZE_32 + NUM_SIZE_32), (void *)x_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time public key
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8),
                (void *)w_h, PK_SIZE_8, hipMemcpyHostToDevice
            ));
 
            Prehash(keepPrehash, data_d, uctxs_d, hashes_d, indices_d);
 

            state = STATE_CONTINUE;
    	    //printf("Prehashed for new block\n");
        }


        CUDA_CALL(hipDeviceSynchronize());
 
         /*     printf(
            "%s Checking solutions for nonces:\n"
            "           0x%016lX -- 0x%016lX\n",
            TimeStamp(&stamp), base, base + THREAD_LEN * LOAD_LEN - 1
        );
        fflush(stdout);
        */   
        // generate nonces
        GenerateConseqNonces<<<1 + (THREAD_LEN * LOAD_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            (uint64_t *)nonces_d, N_LEN, base
        );

        base += THREAD_LEN * LOAD_LEN;
        // calculate unfinalized hash of message
        InitMining(&ctx_h, (uint32_t *)mes_h, NUM_SIZE_8);

        // copy context
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + PK2_SIZE_32 + 3 * NUM_SIZE_32), (void *)&ctx_h,
            sizeof(context_t), hipMemcpyHostToDevice
        ));

        // calculate solution candidates
        BlockMining<<<1 + (LOAD_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            bound_d, data_d, nonces_d, hashes_d, res_d, indices_d
        );

        // try to find solution
        ind = FindNonZero(
            indices_d, indices_d + THREAD_LEN * LOAD_LEN, THREAD_LEN * LOAD_LEN
        );

        // solution found
        if (ind)
        {
            CUDA_CALL(hipMemcpy(
                (void *)res_h, (void *)(res_d + ((ind - 1) << 3)), NUM_SIZE_8,
                hipMemcpyDeviceToHost
            ));

            CUDA_CALL(hipMemcpy(
                (void *)nonces_h, (void *)(nonces_d + ((ind - 1) << 1)),
                NONCE_SIZE_8, hipMemcpyDeviceToHost
            ));

            printf("%s Solution found from GPU %i:\n", TimeStamp(&stamp), deviceId); 
            PrintPuzzleSolution(nonces_h, res_h);
            PostPuzzleSolution(to, pkstr, w_h, nonces_h, res_h);
            printf("new Solution is posted\n");
            fflush(stdout);
	
            state = STATE_KEYGEN;
        }
    }
    while(1); // !TerminationRequestHandler()); 

    return;
}

// autolykos.cu
