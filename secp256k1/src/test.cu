#include "hip/hip_runtime.h"
// test.cu

/*******************************************************************************

    TEST -- hash functions test suite

*******************************************************************************/

#include "../include/cryptography.h"
#include "../include/definitions.h"
#include "../include/easylogging++.h"
#include "../include/mining.h"
#include "../include/prehash.h"
#include "../include/reduction.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <curl/curl.h>
#include <inttypes.h>
#include <iostream>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <atomic>
#include <chrono>
#include <mutex>
#include <thread>

INITIALIZE_EASYLOGGINGPP

namespace cg = cooperative_groups;
namespace ch = std::chrono;

template<uint32_t blockSize>
__global__ void BlockSum(
    uint32_t * in,
    uint32_t inlen,
    uint32_t * out
)
{
    uint32_t ind = 0;
    uint32_t tid = threadIdx.x;
    __shared__ uint32_t sdata[BLOCK_DIM];

    cg::thread_block cta = cg::this_thread_block();

    for (
        uint32_t i = 2 * blockIdx.x * blockSize + tid;
        i < inlen;
        i += 2 * blockSize * gridDim.x
    )
    {
        ind += 1 - !(in[i]);
        ind += !(i + blockSize >= inlen) * (1 - !(in[i + blockSize]));
    }

    sdata[tid] = ind;
    cg::sync(cta);

#if (__CUDA_ARCH__ >= 300)
    if (tid < 32)
    {
        cg::coalesced_group active = cg::coalesced_threads();

        if (blockSize >= 64) { ind += sdata[tid + 32]; }

        for (int offset = warpSize >> 1; offset > 0; offset >>= 1) 
        {
             ind += 1 - !(active.shfl_down(ind, offset));
        }
    }
#else
    if (blockSize >= 64 && tid < 32)
    {
        sdata[tid] = ind = ind + sdata[tid + 32];
    }

    cg::sync(cta);

    if (blockSize >= 32 && tid < 16)
    {
        sdata[tid] = ind = ind + sdata[tid + 16];
    }

    cg::sync(cta);

    if (blockSize >= 16 && tid < 8)
    {
        sdata[tid] = ind = ind + sdata[tid + 8];
    }

    cg::sync(cta);

    if (blockSize >= 8 && tid < 4)
    {
        sdata[tid] = ind = ind + sdata[tid + 4];
    }

    cg::sync(cta);

    if (blockSize >= 4 && tid < 2)
    {
        sdata[tid] = ind = ind + sdata[tid + 2];
    }

    cg::sync(cta);

    if (blockSize >= 2 && tid < 1) { ind += sdata[tid + 1]; }
    cg::sync(cta);
#endif

    if (!tid) { out[blockIdx.x] = ind; }

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Find non zero item in each block of array
////////////////////////////////////////////////////////////////////////////////
void ReduceSum(
    uint32_t * in,
    uint32_t inlen,
    uint32_t * out,
    uint32_t gridSize,
    uint32_t blockSize
)
{
    switch (blockSize)
    {
        case 64:
            BlockSum<64><<<gridSize, blockSize>>>(in, inlen, out);
            break;

        case 32:
            BlockSum<32><<<gridSize, blockSize>>>(in, inlen, out);
            break;

        case 16:
            BlockSum<16><<<gridSize, blockSize>>>(in, inlen, out);
            break;

        case 8:
            BlockSum<8><<<gridSize, blockSize>>>(in, inlen, out);
            break;

        case 4:
            BlockSum<4><<<gridSize, blockSize>>>(in, inlen, out);
            break;

        case 2:
            BlockSum<2><<<gridSize, blockSize>>>(in, inlen, out);
            break;

        case 1:
            BlockSum<1><<<gridSize, blockSize>>>(in, inlen, out);
            break;
    }

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Find non zero item in array
////////////////////////////////////////////////////////////////////////////////
uint32_t FindSum(
    uint32_t * data,
    uint32_t * aux,
    uint32_t inlen
)
{
    uint32_t res;
    uint32_t blockSize = (BLOCK_DIM < 64)? CeilToPower(BLOCK_DIM): 64;
    uint32_t gridSize = 1 + (inlen - 1) / (blockSize << 1);
    uint32_t * tmp;

    while (inlen > 1)
    {
        ReduceSum(data, inlen, aux, gridSize, blockSize);

        inlen = gridSize;

        if (inlen < 64) { blockSize = CeilToPower((inlen + 1) >> 1); }

        gridSize = 1 + (inlen - 1) / (blockSize << 1);

        tmp = data;
        data = aux;
        aux = tmp;
    }

    CUDA_CALL(hipMemcpy(
        &res, data, INDEX_SIZE_8, hipMemcpyDeviceToHost
    ));

    return res;
}

////////////////////////////////////////////////////////////////////////////////
//  Test solutions correctness
////////////////////////////////////////////////////////////////////////////////
int TestSolutions(
    const info_t * info,
    const uint8_t * x,
    const uint8_t * w
)
{
    LOG(INFO) << "Set keepPrehash == "
        << ((info->keepPrehash)? "true": "false");
    LOG(INFO) << "Solutions test started";

    //========================================================================//
    //  Host memory allocation
    //========================================================================//
    // hash context
    // (212 + 4) bytes
    ctx_t ctx_h;

    //========================================================================//
    //  Device memory allocation
    //========================================================================//
    // boundary for puzzle
    // ~0 MiB
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc(&bound_d, NUM_SIZE_8 + DATA_SIZE_8));
    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MiB
    uint32_t * data_d = bound_d + NUM_SIZE_32;

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GiB
    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc(&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // WORKSPACE_SIZE_8 bytes
    // potential solutions of puzzle
    uint32_t * res_d;
    CUDA_CALL(hipMalloc(&res_d, WORKSPACE_SIZE_8));
    // indices of unfinalized hashes
    uint32_t * indices_d = res_d + NONCES_PER_ITER * NUM_SIZE_32;

    uctx_t * uctxs_d = NULL;

    if (info->keepPrehash)
    {
        CUDA_CALL(hipMalloc(&uctxs_d, (uint32_t)N_LEN * sizeof(uctx_t)));
    }

    //========================================================================//
    //  Data transfer form host to device
    //========================================================================//
    // copy boundary
    CUDA_CALL(hipMemcpy(
        bound_d, info->bound, NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    // copy public key
    CUDA_CALL(hipMemcpy(data_d, info->pk, PK_SIZE_8, hipMemcpyHostToDevice));

    // copy message
    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8, info->mes, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // copy one time public key
    CUDA_CALL(hipMemcpy(
        ((uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8), w, PK_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // copy one time secret key
    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + NUM_SIZE_32, x, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // copy secret key
    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 2 * NUM_SIZE_32, info->sk, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    //========================================================================//
    //  Test solutions
    //========================================================================//
    uint64_t base = 0;

    if (info->keepPrehash)
    {
        UncompleteInitPrehash<<<1 + (N_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            data_d, uctxs_d
        );
    }

    Prehash(info->keepPrehash, data_d, uctxs_d, hashes_d, res_d);
    CUDA_CALL(hipDeviceSynchronize());

    // calculate unfinalized hash of message
    InitMining(&ctx_h, (uint32_t *)info->mes, NUM_SIZE_8);

    // copy context
    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 3 * NUM_SIZE_32, &ctx_h, sizeof(ctx_t),
        hipMemcpyHostToDevice
    ));

    // calculate solution candidates
    BlockMining<<<1 + (THREADS_PER_ITER - 1) / BLOCK_DIM, BLOCK_DIM>>>(
        bound_d, data_d, base, hashes_d, res_d, indices_d
    );

    const uint32_t ref_indices[3] = { 0x3381BD, 0x376C26, 0x3D5B84 };

    const uint64_t ref_res[3 * NUM_SIZE_64] = {
        0xA41F6C4914B3BCD0, 0x71EEA8CF5356CF28, 0xADB7E97512C1B9AD,
        0x8081936D54481DD8, 0x661D4798E2309692, 0x7EAE28B576532950,
        0x3D2B0B32A1E52137, 0x2406A4B8304E264A, 0x1329C47EBABBB9A8,
        0x9D7AFFEA975A94CF, 0xABFBCFEA7171F4AA, 0x3BA19A1A3D28B102
    };

    uint64_t res_h[3 * NUM_SIZE_64];

    for (int i = 0; i < 3; ++i)
    {
        // copy results to host
        CUDA_CALL(hipMemcpy(
            res_h, res_d + ref_indices[i] * NUM_SIZE_32, NUM_SIZE_8,
            hipMemcpyDeviceToHost
        ));

        if (memcmp(res_h, ref_res + i * NUM_SIZE_64, NUM_SIZE_8))
        {
            LOG(ERROR) << "Solutions test failed";
            exit(EXIT_FAILURE);
        }
    }

    //========================================================================//
    //  Device memory deallocation
    //========================================================================//
    CUDA_CALL(hipFree(bound_d));
    CUDA_CALL(hipFree(hashes_d));
    CUDA_CALL(hipFree(res_d));

    if (info->keepPrehash) { CUDA_CALL(hipFree(uctxs_d)); }

    LOG(INFO) << "Solutions test passed\n";
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//  Test performance
////////////////////////////////////////////////////////////////////////////////
int TestPerformance(
    const info_t * info,
    const uint8_t * x,
    const uint8_t * w
)
{
    LOG(INFO) << "Set keepPrehash == "
        << ((info->keepPrehash)? "true": "false");
    LOG(INFO) << "Performance test started";

    //========================================================================//
    //  Host memory allocation
    //========================================================================//
    // hash context
    // (212 + 4) bytes
    ctx_t ctx_h;

    //========================================================================//
    //  Device memory allocation
    //========================================================================//
    // boundary for puzzle
    // ~0 MiB
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc(&bound_d, NUM_SIZE_8 + DATA_SIZE_8));
    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MiB
    uint32_t * data_d = bound_d + NUM_SIZE_32;

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GiB
    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc(&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // WORKSPACE_SIZE_8 bytes
    // potential solutions of puzzle
    uint32_t * res_d;
    CUDA_CALL(hipMalloc(&res_d, WORKSPACE_SIZE_8));
    // indices of unfinalized hashes
    uint32_t * indices_d = res_d + NONCES_PER_ITER * NUM_SIZE_32;

    uctx_t * uctxs_d = NULL;

    if (info->keepPrehash)
    {
        CUDA_CALL(hipMalloc(&uctxs_d, (uint32_t)N_LEN * sizeof(uctx_t)));
    }

    //========================================================================//
    //  Data transfer form host to device
    //========================================================================//
    // copy boundary
    CUDA_CALL(hipMemcpy(
        bound_d, info->bound, NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    // copy public key
    CUDA_CALL(hipMemcpy(data_d, info->pk, PK_SIZE_8, hipMemcpyHostToDevice));

    // copy message
    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8, info->mes, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // copy one time public key
    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8, w, PK_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // copy one time secret key
    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + NUM_SIZE_32, x, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    // copy secret key
    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 2 * NUM_SIZE_32, info->sk, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    //========================================================================//
    //  Test solutions
    //========================================================================//
    uint64_t base = 0;

    ch::milliseconds ms = ch::milliseconds::zero(); 
    ch::milliseconds start = ch::duration_cast<ch::milliseconds>(
        ch::system_clock::now().time_since_epoch()
    );

    if (info->keepPrehash)
    {
        UncompleteInitPrehash<<<1 + (N_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            data_d, uctxs_d
        );
    }

    Prehash(info->keepPrehash, data_d, uctxs_d, hashes_d, res_d);
    CUDA_CALL(hipDeviceSynchronize());

    ms += ch::duration_cast<ch::milliseconds>(
        ch::system_clock::now().time_since_epoch()
    ) - start;

    LOG(INFO) << "Prehash time: " << ms.count() << " ms";
    LOG(INFO) << "BlockMining now for 1 munute";
    ms = ch::milliseconds::zero();

    // calculate unfinalized hash of message
    InitMining(&ctx_h, (uint32_t *)info->mes, NUM_SIZE_8);

    // copy context
    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 3 * NUM_SIZE_32, &ctx_h, sizeof(ctx_t),
        hipMemcpyHostToDevice
    ));

    uint32_t sum = 0;

    start = ch::duration_cast<ch::milliseconds>(
        ch::system_clock::now().time_since_epoch()
    );

    while (ms.count() < 60000)
    {
        // calculate solution candidates
        BlockMining<<<1 + (THREADS_PER_ITER - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            bound_d, data_d, base, hashes_d, res_d, indices_d
        );

        sum = FindSum(indices_d, indices_d + NONCES_PER_ITER, NONCES_PER_ITER);

        base += NONCES_PER_ITER;

        ms = ch::duration_cast<ch::milliseconds>(
            ch::system_clock::now().time_since_epoch()
        ) - start;
    }

    //========================================================================//
    //  Device memory deallocation
    //========================================================================//
    CUDA_CALL(hipFree(bound_d));
    CUDA_CALL(hipFree(hashes_d));
    CUDA_CALL(hipFree(res_d));

    if (info->keepPrehash) { CUDA_CALL(hipFree(uctxs_d)); }

    LOG(INFO) << "Performance test completed";
    LOG(INFO) << "Found " << sum << " solutions\n";
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//  Main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    START_EASYLOGGINGPP(argc, argv);

    el::Loggers::reconfigureAllLoggers(
        el::ConfigurationType::Format, "%datetime %level [%thread] %msg"
    );

    el::Helpers::setThreadName("test thread");

    //========================================================================//
    //  Check requirements
    //========================================================================//
    int deviceCount;

    if (hipGetDeviceCount(&deviceCount) != hipSuccess)
    {
        LOG(ERROR) << "Error checking GPU";
        exit(EXIT_FAILURE);
    }

    size_t freeMem;
    size_t totalMem;

    CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
    
    if (freeMem < MIN_FREE_MEMORY)
    {
        LOG(ERROR) << "Not enough GPU memory for mining,"
            << " minimum 2.8 GiB needed";

        exit(EXIT_FAILURE);
    }
    
    //========================================================================//
    //  Set test info
    //========================================================================//
    info_t info;
    uint8_t x[NUM_SIZE_8];
    uint8_t w[PK_SIZE_8];
    char seed[256] = "Va'esse deireadh aep eigean, va'esse eigh faidh'ar";

    // generate secret key from seed
    GenerateSecKey(seed, 50, info.sk, info.skstr);
    // generate public key from secret key
    GeneratePublicKey(info.skstr, info.pkstr, info.pk);

    const char ref_pkstr[PK_SIZE_4 + 1]
        = "020C16DFC5E23C59357E89D44977038F0A7851CC9926B3AABB3FF9E7E6A57315AD";

    int test = !strncmp(ref_pkstr, info.pkstr, PK_SIZE_4);

    if (!test)
    {
        LOG(ERROR) << "OpenSSL: generated wrong public key";
        return EXIT_FAILURE;
    }

    ((uint64_t *)info.bound)[0] = 0xFFFFFFFFFFFFFFFF;
    ((uint64_t *)info.bound)[1] = 0xFFFFFFFFFFFFFFFF;
    ((uint64_t *)info.bound)[2] = 0xFFFFFFFFFFFFFFFF;
    ((uint64_t *)info.bound)[3] = 0x00000FFFFFFFFFFF;

    ((uint64_t *)info.mes)[0] = 1;
    ((uint64_t *)info.mes)[1] = 0;
    ((uint64_t *)info.mes)[2] = 0;
    ((uint64_t *)info.mes)[3] = 0;

    sprintf(seed, "%d", 0);

    // generate secret key from seed
    GenerateSecKey(seed, 1, x, info.skstr);
    // generate public key from secret key
    GeneratePublicKey(info.skstr, info.pkstr, w);

    //========================================================================//
    //  Run solutions correctness tests
    //========================================================================//
    if (NONCES_PER_ITER <= 0x3D5B84)
    {
        LOG(INFO) << "Need WORKSPACE value for at least 4021125,"
            << " skip solutions tests\n";
    }
    else
    {
        info.keepPrehash = 0;
        TestSolutions(&info, x, w);

        if (freeMem < MIN_FREE_MEMORY_PREHASH)
        {
            LOG(INFO) << "Not enough GPU memory for keeping prehashes, "
                << "skip test\n";
        }
        else
        {
            info.keepPrehash = 1;
            TestSolutions(&info, x, w);
        }
    }

    //========================================================================//
    //  Run performance tests
    //========================================================================//
    info.keepPrehash = 0;
    TestPerformance(&info, x, w);

    if (freeMem >= MIN_FREE_MEMORY_PREHASH)
    {
        info.keepPrehash = 1;
        TestPerformance(&info, x, w);
    }

    LOG(INFO) << "All tests have been successfully completed";

    return EXIT_SUCCESS;
}

// test.cu
