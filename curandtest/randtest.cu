#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>


#define CUDA_CALL(x) do { if((x) != hipSuccess) {  \
printf("Error at %s:%d\n",__FILE__,__LINE__);       \
return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) {  \
printf("Error at %s:%d\n",__FILE__,__LINE__);                   \
return EXIT_FAILURE;}} while(0)

__global__ void generate_kernel(
    hiprandStateMtgp32 * state, 
    int n,
    int *result
) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int count = 0;
    unsigned int x;
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&state[blockIdx.x]);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{
    int blockSize;   // The launch configurator returned block size 
    int minGridSize; // The minimum grid size needed to achieve the 
    // maximum occupancy for a full device launch 

    hipOccupancyMaxPotentialBlockSize(
        &minGridSize, &blockSize, generate_kernel, 0, 0
    ); 

    hipDeviceSynchronize(); 

    printf("grs = %d, bls = %d\n", minGridSize, blockSize);
    ///
    /// // calculate theoretical occupancy
    /// int maxActiveBlocks;
    /// cudaOccupancyMaxActiveBlocksPerMultiprocessor(
    ///     &maxActiveBlocks, MyKernel, blockSize, 0
    /// );

    /// int device;
    /// cudaDeviceProp props;
    /// cudaGetDevice(&device);
    /// cudaGetDeviceProperties(&props, device);

    /// float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
    ///     (float)(props.maxThreadsPerMultiProcessor / props.warpSize);

    /// printf(
    ///     "Launched blocks of size %d. Theoretical occupancy: %f\n",
    ///     blockSize, occupancy
    /// );

    int i;
    long long total;
    hiprandStateMtgp32 * devMTGPStates;
    mtgp32_kernel_params * devKernelParams;

    int * devResults, * hostResults;
    int sampleCount = 10000;
    
    /* Allow over-ride of sample count */    
    if (argc == 2) {
        sscanf(argv[1],"%d",&sampleCount);
    }
        
    /* Allocate space for results on host */
    hostResults = (int *)calloc(64 * 512, sizeof(int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 512 * sizeof(int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 512 * sizeof(int)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, 64 * sizeof(hiprandStateMtgp32)));
    
    ////////////////////////////////////////////////////////////////////////
    /* Allocate space for MTGP kernel parameters */
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
    
    /* Reformat from predefined parameter sets to kernel format, */
    /* and copy kernel parameters to device memory               */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
    
    /* Initialize one state per thread block */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, 
                mtgp32dc_params_fast_11213, devKernelParams, 64, 1234));
    ////////////////////////////////////////////////////////////////////////
    
    /* Generate and use pseudo-random  */
    for(i = 0; i < 10; i++) {
        generate_kernel<<<64, 512>>>(devMTGPStates, sampleCount, devResults);
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(
        hostResults, devResults, 64 * 512 * sizeof(int), hipMemcpyDeviceToHost)
    );

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 512; i++) {
        total += hostResults[i];
    }
    
    printf("Fraction with low bit set was %10.13g\n", 
        (double)total / (64.0f * 512.0f * sampleCount * 10.0f));

    /* Cleanup */
    CUDA_CALL(hipFree(devMTGPStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);

    printf("OK\n");
    return EXIT_SUCCESS;
}

