#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <inttypes.h>

// 8 * 32 bits
// little-endian
// q == [q0, q1, q2, q3, 0, 0, 0, 0x10000000]

// 32 bits
#define q3_Curve25519_s "0x14DEF9DE"
#define q2_Curve25519_s "0xA2F79CD6"
#define q1_Curve25519_s "0x5812631A"
#define q0_Curve25519_s "0x5CF5D3ED"

#define q3_secp256k1_s "0xBAAEDCE6"
#define q2_secp256k1_s "0xAF48A03B"
#define q1_secp256k1_s "0xBFD25E8C"
#define q0_secp256k1_s "0xD0364141"

// 64 bits
#define Q1_secp256k1 0xBAAEDCE6AF48A03B
#define Q0_secp256k1 0xBFD25E8CD0364141

////////////////////////////////////////////////////////////////////////////////
//  256 bits addition with overflow
////////////////////////////////////////////////////////////////////////////////
// carry must be initialized
// [x, carry] + y -> [x, carry]
__global__ void addc(
    // 8 * 32 bits
    uint32_t * x,
    // 1 * 32 bits
    uint32_t * carry,
    // 8 * 32 bits
    uint32_t * y
) {
    asm volatile (
        "add.cc.u32 %0, %0, %1;": "+r"(x[0]): "r"(y[0])
    );

#pragma unroll
    for (int i = 1; i < 8; ++i)
    {
        asm volatile (
            "addc.cc.u32 %0, %0, %1;": "+r"(x[i]): "r"(y[i])
        );
    }

    asm volatile (
        "addc.u32 %0, %0, 0;": "+r"(*carry)
    );
}

////////////////////////////////////////////////////////////////////////////////
//  256 bits subtraction with borrow-out
////////////////////////////////////////////////////////////////////////////////
// carry must be initialized
// [x, carry] - y -> [x, carry]
__global__ void subc(
    // 8 * 32 bits
    uint32_t * x,
    // 1 * 32 bits
    uint32_t * carry,
    // 8 * 32 bits
    uint32_t * y
) {
    asm volatile (
        "sub.cc.u32 %0, %0, %1;": "+r"(x[0]): "r"(y[0])
    );

#pragma unroll
    for (int i = 1; i < 8; ++i)
    {
        asm volatile (
            "subc.cc.u32 %0, %0, %1;": "+r"(x[i]): "r"(y[i])
        );
    }

    asm volatile (
        "subc.u32 %0, %0, 0;": "+r"(*carry)
    );
}

////////////////////////////////////////////////////////////////////////////////
//  256 bits multiplication
////////////////////////////////////////////////////////////////////////////////
__global__ void mul(
    // 8 * 32 bits
    uint32_t * x,
    // 8 * 32 bits
    uint32_t * y,
    // 16 * 32 bits
    uint32_t * res
) {
    //====================================================================//
    //  x[0] * y -> res[0, ..., 7, 8]
    //====================================================================//
    // initialize res[0, ..., 7]
#pragma unroll
    for (int j = 0; j < 8; j += 2)
    {
        asm volatile (
            "mul.lo.u32 %0, %1, %2;": "=r"(res[j]): "r"(x[0]), "r"(y[j])
        );
        asm volatile (
            "mul.hi.u32 %0, %1, %2;": "=r"(res[j + 1]): "r"(x[0]), "r"(y[j])
        );
    }

    //====================================================================//
    asm volatile (
        "mad.lo.cc.u32 %0, %1, %2, %0;": "+r"(res[1]): "r"(x[0]), "r"(y[1])
    );
    asm volatile (
        "madc.hi.cc.u32 %0, %1, %2, %0;": "+r"(res[2]): "r"(x[0]), "r"(y[1])
    );

#pragma unroll
    for (int j = 3; j < 6; j += 2)
    {
        asm volatile (
            "madc.lo.cc.u32 %0, %1, %2, %0;": "+r"(res[j]): "r"(x[0]), "r"(y[j])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, %2, %0;":
            "+r"(res[j + 1]): "r"(x[0]), "r"(y[j])
        );
    }

    asm volatile (
        "madc.lo.cc.u32 %0, %1, %2, %0;": "+r"(res[7]): "r"(x[0]), "r"(y[7])
    );
    // initialize res[8]
    asm volatile (
        "madc.hi.u32 %0, %1, %2, 0;": "=r"(res[8]): "r"(x[0]), "r"(y[7])
    );

    //====================================================================//
    //  x[i] * y -> res[i, ..., i + 7, i + 8]
    //====================================================================//
#pragma unroll
    for (int i = 1; i < 8; ++i)
    {
        asm volatile (
            "mad.lo.cc.u32 %0, %1, %2, %0;": "+r"(res[i]): "r"(x[i]), "r"(y[0])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, %2, %0;":
            "+r"(res[i + 1]): "r"(x[i]), "r"(y[0])
        );

#pragma unroll
        for (int j = 2; j < 8; j += 2)
        {
            asm volatile (
                "madc.lo.cc.u32 %0, %1, %2, %0;":
                "+r"(res[i + j]): "r"(x[i]), "r"(y[j])
            );
            asm volatile (
                "madc.hi.cc.u32 %0, %1, %2, %0;":
                "+r"(res[i + j + 1]): "r"(x[i]), "r"(y[j])
            );
        }

    // initialize res[i + 8]
        asm volatile (
            "addc.u32 %0, 0, 0;": "=r"(res[i + 8])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, %2, %0;":
            "+r"(res[i + 1]): "r"(x[i]), "r"(y[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, %2, %0;":
            "+r"(res[i + 2]): "r"(x[i]), "r"(y[1])
        );

#pragma unroll
        for (int j = 3; j < 6; j += 2)
        {
            asm volatile (
                "madc.lo.cc.u32 %0, %1, %2, %0;":
                "+r"(res[i + j]): "r"(x[i]), "r"(y[j])
            );
            asm volatile (
                "madc.hi.cc.u32 %0, %1, %2, %0;":
                "+r"(res[i + j + 1]): "r"(x[i]), "r"(y[j])
            );
        }

        asm volatile (
            "madc.lo.cc.u32 %0, %1, %2, %0;":
            "+r"(res[i + 7]): "r"(x[i]), "r"(y[7])
        );
        asm volatile (
            "madc.hi.u32 %0, %1, %2, %0;":
            "+r"(res[i + 8]): "r"(x[i]), "r"(y[7])
        );
    }
}

////////////////////////////////////////////////////////////////////////////////
//  Mod q
////////////////////////////////////////////////////////////////////////////////
__global__ void mod_q_Curve25519(
    // word count
    uint32_t xw,
    // xw * 64 bits
    uint64_t * x
    // result 4 * 64 bits -> x[0, 1, 2, 3]
) {
    uint32_t * y = (uint32_t *)x; 
    uint32_t d[2]; 
    uint32_t med[6];
    uint32_t carry;

    for (int i = (xw - 1) << 1; i >= 8; i -= 2)
    {
        *((uint64_t *)d) = ((x[i >> 1] << 4) | (x[(i >> 1) - 1] >> 60))
            - (x[i >> 1] >> 60);

        // correct highest 32 bits
        y[i - 1] = (y[i - 1] & 0x0FFFFFFF) | y[i + 1] & 0x10000000;

    //====================================================================//
    //  d * q -> med[0, ..., 5]
    //====================================================================//
        asm volatile (
            "mul.lo.u32 %0, %1, "q0_Curve25519_s";": "=r"(med[0]): "r"(d[0])
        );
        asm volatile (
            "mul.hi.u32 %0, %1, "q0_Curve25519_s";": "=r"(med[1]): "r"(d[0])
        );
        asm volatile (
            "mul.lo.u32 %0, %1, "q2_Curve25519_s";": "=r"(med[2]): "r"(d[0])
        );
        asm volatile (
            "mul.hi.u32 %0, %1, "q2_Curve25519_s";": "=r"(med[3]): "r"(d[0])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q1_Curve25519_s", %0;": "+r"(med[1]): "r"(d[0])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q1_Curve25519_s", %0;": "+r"(med[2]): "r"(d[0])
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_Curve25519_s", %0;": "+r"(med[3]): "r"(d[0])
        );
        asm volatile (
            "madc.hi.u32 %0, %1, "q3_Curve25519_s", 0;": "=r"(med[4]): "r"(d[0])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q0_Curve25519_s", %0;": "+r"(med[1]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q0_Curve25519_s", %0;": "+r"(med[2]): "r"(d[1])
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q2_Curve25519_s", %0;": "+r"(med[3]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1," q2_Curve25519_s", %0;": "+r"(med[4]): "r"(d[1])
        );
        asm volatile (
            "addc.u32 %0, 0, 0;": "=r"(med[5])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q1_Curve25519_s", %0;": "+r"(med[2]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q1_Curve25519_s", %0;": "+r"(med[3]): "r"(d[1])
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_Curve25519_s", %0;": "+r"(med[4]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.u32 %0, %1, "q3_Curve25519_s", %0;": "+r"(med[5]): "r"(d[1])
        );

    //====================================================================//
    //  x[i/2 - 2, i/2 - 3, i/2 - 4] mod q
    //====================================================================//
        asm volatile (
            "sub.cc.u32 %0, %0, %1;": "+r"(y[i - 8]): "r"(med[0])
        );

#pragma unroll
        for (int j = 1; j < 6; ++j)
        {
            asm volatile (
                "subc.cc.u32 %0, %0, %1;": "+r"(y[i + j - 8]): "r"(med[j])
            );
        }

        asm volatile (
            "subc.cc.u32 %0, %0, 0;": "+r"(y[i - 2])
        );

        asm volatile (
            "subc.cc.u32 %0, %0, 0;": "+r"(y[i - 1])
        );

    //====================================================================//
    //  x[i/2 - 2, i/2 - 3, i/2 - 4] correction
    //====================================================================//
        asm volatile (
            "subc.u32 %0, 0, 0;": "=r"(carry)
        );

        carry = 0 - carry;

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q0_Curve25519_s", %0;": "+r"(y[i - 8]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q1_Curve25519_s", %0;": "+r"(y[i - 7]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q2_Curve25519_s", %0;": "+r"(y[i - 6]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_Curve25519_s", %0;": "+r"(y[i - 5]): "r"(carry)
        );

    //====================================================================//
#pragma unroll
        for (int j = 0; j < 3; ++j)
        {
            asm volatile (
                "addc.cc.u32 %0, %0, 0;": "+r"(y[i + j - 4])
            );
        }

        asm volatile (
            "addc.u32 %0, %0, 0;": "+r"(y[i - 1])
        );
    }
}

////////////////////////////////////////////////////////////////////////////////
//  Mod q secp256k1
////////////////////////////////////////////////////////////////////////////////
__global__ void mod_q_secp256k1(
    // word count
    uint32_t xw,
    // xw * 64 bits
    uint64_t * x
    // result 4 * 64 bits -> x[0, 1, 2, 3]
) {
    uint32_t * y = (uint32_t *)x; 
    uint32_t d[2]; 
    uint32_t med[6];
    uint32_t carry;

    for (int i = (xw - 1) << 1; i >= 8; i -= 2)
    {
        *((uint64_t *)d) = x[i >> 1];

    //====================================================================//
    //  med[0, ..., 5] = d * q
    //====================================================================//
        asm volatile (
            "mul.lo.u32 %0, %1, "q0_secp256k1_s";": "=r"(med[0]): "r"(d[0])
        );
        asm volatile (
            "mul.hi.u32 %0, %1, "q0_secp256k1_s";": "=r"(med[1]): "r"(d[0])
        );
        asm volatile (
            "mul.lo.u32 %0, %1, "q2_secp256k1_s";": "=r"(med[2]): "r"(d[0])
        );
        asm volatile (
            "mul.hi.u32 %0, %1, "q2_secp256k1_s";": "=r"(med[3]): "r"(d[0])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q1_secp256k1_s", %0;": "+r"(med[1]): "r"(d[0])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q1_secp256k1_s", %0;": "+r"(med[2]): "r"(d[0])
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_secp256k1_s", %0;": "+r"(med[3]): "r"(d[0])
        );
        asm volatile (
            "madc.hi.u32 %0, %1, "q3_secp256k1_s", 0;": "=r"(med[4]): "r"(d[0])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q0_secp256k1_s", %0;": "+r"(med[1]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q0_secp256k1_s", %0;": "+r"(med[2]): "r"(d[1])
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q2_secp256k1_s", %0;": "+r"(med[3]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1," q2_secp256k1_s", %0;": "+r"(med[4]): "r"(d[1])
        );
        asm volatile (
            "addc.u32 %0, 0, 0;": "=r"(med[5])
        );

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q1_secp256k1_s", %0;": "+r"(med[2]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.cc.u32 %0, %1, "q1_secp256k1_s", %0;": "+r"(med[3]): "r"(d[1])
        );
        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_secp256k1_s", %0;": "+r"(med[4]): "r"(d[1])
        );
        asm volatile (
            "madc.hi.u32 %0, %1, "q3_secp256k1_s", %0;": "+r"(med[5]): "r"(d[1])
        );

    //====================================================================//
    //  x[i/2 - 1, i/2 - 2] += 2 * d
    //====================================================================//
        carry = d[1] >> 31;
        *((uint64_t *)d) <<= 1;

        asm volatile (
            "add.cc.u32 %0, %0, %1;": "+r"(y[i - 4]): "r"(d[0])
        );

        asm volatile (
            "addc.cc.u32 %0, %0, %1;": "+r"(y[i - 3]): "r"(d[1])
        );

        asm volatile (
            "addc.cc.u32 %0, %0, %1;": "+r"(y[i - 2]): "r"(carry)
        );

        asm volatile (
            "addc.cc.u32 %0, %0, 0;": "+r"(y[i - 1])
        );

        asm volatile (
            "addc.u32 %0, 0, 0;": "=r"(carry)
        );

    //====================================================================//
    //  x[i/2 - 2, i/2 - 3, i/2 - 4] mod q
    //====================================================================//
        asm volatile (
            "sub.cc.u32 %0, %0, %1;": "+r"(y[i - 8]): "r"(med[0])
        );

#pragma unroll
        for (int j = 1; j < 6; ++j)
        {
            asm volatile (
                "subc.cc.u32 %0, %0, %1;": "+r"(y[i + j - 8]): "r"(med[j])
            );
        }

        asm volatile (
            "subc.cc.u32 %0, %0, 0;": "+r"(y[i - 2])
        );

        asm volatile (
            "subc.cc.u32 %0, %0, 0;": "+r"(y[i - 1])
        );

    //====================================================================//
    //  x[i/2 - 2, i/2 - 3, i/2 - 4] correction
    //====================================================================//
        asm volatile (
            "subc.u32 %0, %0, 0;": "+r"(carry)
        );

        carry = 0 - carry;

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, "q0_secp256k1_s", %0;": "+r"(y[i - 8]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q1_secp256k1_s", %0;": "+r"(y[i - 7]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q2_secp256k1_s", %0;": "+r"(y[i - 6]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, "q3_secp256k1_s", %0;": "+r"(y[i - 5]): "r"(carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xFFFFFFFE, %0;": "+r"(y[i - 4]): "r"(carry)
        );

#pragma unroll
        for (int j = 0; j < 2; ++j)
        {
            asm volatile (
                "madc.lo.cc.u32 %0, %1, 0xFFFFFFFF, %0;": "+r"(y[i + j - 3]): "r"(carry)
            );
        }

        asm volatile (
            "madc.lo.u32 %0, %1, 0xFFFFFFFF, %0;": "+r"(y[i - 1]): "r"(carry)
        );
    }

    //====================================================================//
    //  last correction
    //====================================================================//
    asm volatile (
        "sub.cc.u32 %0, %0, "q0_secp256k1_s";": "+r"(y[0])
    );

    asm volatile (
        "subc.cc.u32 %0, %0, "q1_secp256k1_s";": "+r"(y[1])
    );

    asm volatile (
        "subc.cc.u32 %0, %0, "q2_secp256k1_s";": "+r"(y[2])
    );

    asm volatile (
        "subc.cc.u32 %0, %0, "q3_secp256k1_s";": "+r"(y[3])
    );

    asm volatile (
        "subc.cc.u32 %0, %0, 0xFFFFFFFE;": "+r"(y[4])
    );

#pragma unroll
    for (int j = 5; j < 8; ++j)
    {
        asm volatile (
            "subc.cc.u32 %0, %0, 0xFFFFFFFF;": "+r"(y[j])
        );
    }

    //====================================================================//
    asm volatile (
        "subc.u32 %0, 0, 0;": "=r"(carry)
    );

    carry = 0 - carry;

    //====================================================================//
    asm volatile (
        "mad.lo.cc.u32 %0, %1, "q0_secp256k1_s", %0;": "+r"(y[0]): "r"(carry)
    );

    asm volatile (
        "madc.lo.cc.u32 %0, %1, "q1_secp256k1_s", %0;": "+r"(y[1]): "r"(carry)
    );

    asm volatile (
        "madc.lo.cc.u32 %0, %1, "q2_secp256k1_s", %0;": "+r"(y[2]): "r"(carry)
    );

    asm volatile (
        "madc.lo.cc.u32 %0, %1, "q3_secp256k1_s", %0;": "+r"(y[3]): "r"(carry)
    );

    asm volatile (
        "madc.lo.cc.u32 %0, %1, 0xFFFFFFFE, %0;": "+r"(y[4]): "r"(carry)
    );

#pragma unroll
    for (int j = 5; j < 7; ++j)
    {
        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xFFFFFFFF, %0;": "+r"(y[j]): "r"(carry)
        );
    }

    asm volatile (
        "madc.lo.u32 %0, %1, 0xFFFFFFFF, %0;": "+r"(y[7]): "r"(carry)
    );

    return;
}
